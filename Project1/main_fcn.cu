#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>
#include <sys/time.h>

using namespace std;

//global variables
const bool allow_interrupt = 0;
const int N = 1;
const int numElems =16;

struct help_input_from_main{
	static const int length = N;
	double inp1[N];

	void initS(double* v1){
		int i = 0;
		for(i = 0; i < N; i++){
			inp1[i] = v1[i];
		}
	}

};

struct ctrl_flags{
	bool main_done_cmd = 0;
	bool call_help_cmd = 0;
	volatile bool help_rdy_cmd = 0;
	volatile bool help_running_cmd = 0;
	volatile bool interrupt_help_cmd = 0;
	volatile bool request_val_cmd = 0;
	volatile bool req_delivered_cmd = 0;
	
	bool *call_help = &call_help_cmd;
	volatile bool *help_rdy = &help_rdy_cmd;
	bool *main_done = &main_done_cmd;
	volatile bool *help_running = &help_running_cmd;
	volatile bool *interrupt_help = &interrupt_help_cmd;
	volatile bool *request_val = &request_val_cmd;
	volatile bool *request_done = &req_delivered_cmd;};

//function declarations -- helper and main
bool main_fcn(ctrl_flags CF, double * out_data, help_input_from_main * help);
bool help_fcn(help_input_from_main help_input, double* out);
bool init_help(help_input_from_main help_input);

//function declarations -- calc kernel and monitor kernel
__global__ void dataKernel( double* data,  int nsteps);
__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr);


int main()
{
	//define booleans needed for logic
	ctrl_flags CF;


	//define interface between helper and main i.e.: what is returned
	//double out_val =0.0;


	double out[numElems];

	help_input_from_main test_input;	
	help_input_from_main* help_input = &test_input;

	static double inp1[N] = {5};


	(*help_input).initS(&inp1[0]);	


	#pragma omp parallel num_threads(2) shared(CF, help_input, out)
	{

		if(omp_get_thread_num() == 0){
			cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
		//code for master threads
			CF.main_done_cmd = main_fcn(CF, out, help_input);
		}

		if(omp_get_thread_num() == 1){
			cout <<"Running CUDA init" << endl;

			double hostArray[numElems];
			double *dArray;

			int i = 0;

			//pointer of helper function return	

			double* h_data;
			double* monitor_data;
			

			

			hipMalloc((void**)&dArray, sizeof(double)*numElems);
			//cudaMalloc((void**)&dArray_Held, sizeof(int)*numElems);
			hipMemset(dArray, 0, numElems*sizeof(double));
			//cudaMemset(dArray_Held, 0, numElems*sizeof(int));
			hipMalloc((void**)&monitor_data, sizeof(double)*numElems);
			hipHostMalloc((void**)&h_data, sizeof(double)*numElems, hipHostMallocDefault);
			hipStream_t stream1;
			hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
			//cudaStreamCreate(&stream1);


			while(CF.main_done_cmd == 0){

				if(CF.call_help_cmd == 1 && CF.help_running_cmd == 0){
					CF.help_running_cmd = 1;
					CF.call_help_cmd = 0;
					cout <<"Launching Helper Kernel" << endl;
					//*help_rdy =  help_fcn(*help_input, out);
					dataKernel<<<1,numElems>>>(dArray, 1000);
				}
				if(CF.help_running_cmd == 1 && allow_interrupt == 0 && CF.request_val_cmd == 1){	
					cout <<"Launching Monitor Kernel" << endl;
					//cudaStreamSynchronize(stream1);
					monitorKernel<<<1, numElems,0, stream1>>>(monitor_data, dArray);
					cout <<"Launching Async Mem Cpy" << endl;
					hipMemcpyAsync(h_data, monitor_data, numElems*sizeof(double), hipMemcpyDeviceToHost, stream1);
					hipStreamSynchronize(stream1);
					CF.request_val_cmd = 0;
					for(i = 0; i < numElems; i++)
						out[i] = h_data[i];
					CF.req_delivered_cmd = 1;
				}	
			}


			hipMemcpy(h_data, dArray, sizeof(double)*numElems, hipMemcpyDeviceToHost);
			for(i = 0; i < 5; i++)
				cout << "Value copied over: "  << h_data[i] << endl;

			hipFree(dArray);
			hipFree(monitor_data);
		
	
		}



	}


	return 0;

}

bool main_fcn(ctrl_flags CF, double* help_out, help_input_from_main* help_input_ptr)
{	
	struct timeval stop, start;
	bool *call_help = CF.call_help;
	//volatile bool *help_rdy = CF.help_rdy;
	volatile bool *request_val = CF.request_val;
	volatile bool *request_done = CF.request_done;

	//initialize data for input to helper function
	double inp1[N] = {4};
	int i = 0;
	int numReads = 10;
	double sval;
	double sum_times = 0;
	int j = 0;

	//set values of helper function input
	(*help_input_ptr).initS(inp1);
	//ask to start help function	
	cout << "Main calling help function for 1st time" << endl;
	*call_help = 1;
	
	//=====USER CODE before calling help GOES HERE==========
	sleep(.01);


	for(j = 0; j < numReads; j++){
	gettimeofday(&start, NULL);

		//BELOW IS WHERE YOU CALL THE HELPER READ FROM MAIN
		*request_val = 1;
		while(*request_done == 0)
			sleep(.00000001);
		//ABOVE IS WHERE YOU CALL THE HELPER READ FROM MAIN -- now help value(s) is in *help_out

	gettimeofday(&stop, NULL);
	sval = (stop.tv_sec-start.tv_sec)*1000000; //sec to us
	sval = sval + stop.tv_usec-start.tv_usec; //us

	cout << "Time between message request and message receive in us is: " << sval << endl;
	for(i = 0; i < 3; i++)
		cout << "Main update received " << help_out[i] << endl;
	*request_done = 0;
	if(j > 0)  //skip the first call because its bad fro some reason
		sum_times = sum_times+sval;
	sleep(.2);

	}

	
	cout << "Average read time between message request and message received in us is: " << sum_times/(numReads-1) << endl;

	//=======USER code AFTER calling helper goes here======

	cout << "Exiting Main" << endl;
	
	return 1;


}

bool help_fcn(help_input_from_main help_input, double* out){
	//int j = 1;
	int i = 0;
	double* inp1 = help_input.inp1;
	double* inp2 = help_input.inp1;
	
	for(i = 0; i < N; i++){

		if(i > 0)
			//*out = (*out+inp1[i]+inp2[i])*i/(i+1);
			*out = (*out+inp1[i]+inp2[i]);
		else
			//*out = (*out+inp1[i]+inp2[i]);
			*out = (*out+inp1[i]+inp2[i]);	
		//cout << "out after update = " << *out << endl;	

	
	}
	return 1;
}





__global__ void dataKernel( double* data, int nsteps){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	//data[thid] = 0;
	int i = 0;
	//bool wait = 1;

	//clock_t start = clock64();
	//clock_t now;

	while(i < nsteps){
		data[thid] = data[thid]+.1;
	}

	/*	clock_t start = clock64();
		i = i+1;
		while(wait == 1){
			now = clock();
			clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
			if(cycles > 5000)
				wait = 0;
		}		
		wait = 1;
		
	}	*/



}


__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr){

	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	write_2_ptr[thid] = read_in_ptr[thid];


}

