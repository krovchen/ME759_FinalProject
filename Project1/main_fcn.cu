#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>

using namespace std;

//global variables
const bool allow_interrupt = 0;
const int N = 5;


struct help_input_from_main{
	static const int length = N;
	double inp1[N];
	double inp2[N];

	void initS(double* v1, double* v2){
		int i = 0;
		for(i = 0; i < N; i++){
			inp1[i] = v1[i];
			inp2[i] = v2[i];
		}
	}

};

struct ctrl_flags{
	bool main_done_cmd = 0;
	bool call_help_cmd = 0;
	volatile bool help_rdy_cmd = 0;
	volatile bool help_running_cmd = 0;
	volatile bool interrupt_help_cmd = 0;
	volatile bool request_val_cmd = 0;
	volatile bool req_delivered_cmd = 0;
	
	bool *call_help = &call_help_cmd;
	volatile bool *help_rdy = &help_rdy_cmd;
	bool *main_done = &main_done_cmd;
	volatile bool *help_running = &help_running_cmd;
	volatile bool *interrupt_help = &interrupt_help_cmd;
	volatile bool *request_val = &request_val_cmd;
	volatile bool *request_done = &req_delivered_cmd;};

//function declarations -- helper and main
bool main_fcn(ctrl_flags CF, double * out_data, help_input_from_main* help_input);
bool help_fcn(help_input_from_main help_input, double* out);
bool init_help(help_input_from_main help_input);

//function declarations -- calc kernel and monitor kernel
__global__ void dataKernel( double* data,  int nsteps);
__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr);


int main()
{
	//define booleans needed for logic
	ctrl_flags CF;

	//define interface between helper and main i.e.: what is returned
	double out_val =0.0;


	double *out = &out_val;

	help_input_from_main test_input;	
	help_input_from_main* help_input = &test_input;

	static double inp1[N] = {1,2,3,4,5};
	static double inp2[N] = {1,2,3,4,5};

	(*help_input).initS(&inp1[0], &inp2[0]);	


	#pragma omp parallel num_threads(2) shared(CF)
	{

		if(omp_get_thread_num() == 0){
			cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
		//code for master threads
			CF.main_done_cmd = main_fcn(CF, out, help_input);
		}

		if(omp_get_thread_num() == 1){
			cout <<"Running CUDA init" << endl;
			const int numElems = 1;
			double hostArray[numElems];
			double *dArray;

			int i = 0;

			//pointer of helper function return	

			double *h_data;
			double *monitor_data;

		

			hipMalloc((void**)&dArray, sizeof(double)*numElems);
			//cudaMalloc((void**)&dArray_Held, sizeof(int)*numElems);
			hipMemset(dArray, 0, numElems*sizeof(double));
			//cudaMemset(dArray_Held, 0, numElems*sizeof(int));
			hipMalloc((void**)&monitor_data, sizeof(double));
			hipHostMalloc((void**)&h_data, sizeof(double), hipHostMallocDefault);
			hipStream_t stream1;
			hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);


			while(CF.main_done_cmd == 0){

				if(CF.call_help_cmd == 1 && CF.help_running_cmd == 0){
					CF.help_running_cmd = 1;
					CF.call_help_cmd = 0;
					cout <<"Launching Helper Kernel" << endl;
					//*help_rdy =  help_fcn(*help_input, out);
					dataKernel<<<1,1>>>(dArray, 100);
				}
				if(CF.help_running_cmd == 1 && allow_interrupt == 0 && CF.request_val_cmd == 1){	
					cout <<"Launching Monitor Kernel" << endl;
					//cudaStreamSynchronize(stream1);
					monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dArray);
					cout <<"Launching Async Mem Cpy" << endl;
					hipMemcpyAsync(h_data, monitor_data, sizeof(double), hipMemcpyDeviceToHost, stream1);
					hipStreamSynchronize(stream1);
					CF.request_val_cmd = 0;
					*out = *h_data;
					CF.req_delivered_cmd = 1;
				}	
			}


			hipMemcpy(h_data, dArray, sizeof(double), hipMemcpyDeviceToHost);
			cout << "Value copied over: "  << *h_data << endl;

			hipFree(dArray);
			hipFree(monitor_data);
		
	
		}



	}


	return 0;

}

bool main_fcn(ctrl_flags CF, double* help_out, help_input_from_main* help_input_ptr)
{	
	bool *call_help = CF.call_help;
	//volatile bool *help_rdy = CF.help_rdy;
	volatile bool *request_val = CF.request_val;
	volatile bool *request_done = CF.request_done;

	//initialize data for input to helper function
	double inp1[N] = {1,2,3,4,5};
	

	//set values of helper function input
	(*help_input_ptr).initS(inp1, inp1);
	//ask to start help function	
	cout << "Main calling help function for 1st time" << endl;
	*call_help = 1;
	
	//some code/processing goes here
	sleep(1);

	//if interrupt not allowed, then request value from help
	if(allow_interrupt == 0){	
		cout << "Main requesting function update" << endl;
		*request_val = 1;
		while(*request_done == 0)
			sleep(1);
	}


	cout << "Main update received " << *help_out << endl;
	*request_done = 0;
	sleep(2);

	cout << "Main Requestiong Second function update " << endl;
	cout << "Current Request Val (shoudl be 0) = " << *request_val << endl;
	
	*request_val = 1;
	while(*request_done == 0)
		sleep(1);
	cout << "Main update received " << *help_out << endl;
	*request_done = 0;
	sleep(2);

	cout << "Main Requestiong Third function update " << endl;
	*request_val = 1;
	while(*request_done == 0)
		sleep(1);
	cout << "Main update received " << *help_out << endl;
	*request_done = 0;
	//sleep(2);
	cout << "Exiting Main" << endl;
	
	return 1;


}

bool help_fcn(help_input_from_main help_input, double* out){
	//int j = 1;
	int i = 0;
	double* inp1 = help_input.inp1;
	double* inp2 = help_input.inp2;
	
	for(i = 0; i < N; i++){

		if(i > 0)
			//*out = (*out+inp1[i]+inp2[i])*i/(i+1);
			*out = (*out+inp1[i]+inp2[i]);
		else
			//*out = (*out+inp1[i]+inp2[i]);
			*out = (*out+inp1[i]+inp2[i]);	
		//cout << "out after update = " << *out << endl;	

	
	}
	return 1;
}

bool init_help(help_input_from_main help_input){
	

return 1;

}



__global__ void dataKernel( double* data, int nsteps){
//this adds a value to a variable stored in global memory
	*data = 0;
	int i = 0;
	bool wait = 1;

	clock_t start = clock64();
	clock_t now;

	while(i < nsteps){
		*data = *data+.1;

		clock_t start = clock64();
		i = i+1;
		while(wait == 1){
			now = clock();
			clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
			if(cycles > 5)
				wait = 0;
		}		
		wait = 1;
	}	



}


__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr){

	*write_2_ptr = *read_in_ptr;


}

