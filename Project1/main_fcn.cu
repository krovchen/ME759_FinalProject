#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>

using namespace std;

//global variables
const bool allow_interrupt = 0;
const int N = 5;

struct help_input_from_main{
	static const int length = N;
	double inp1[N];
	double inp2[N];

	void initS(const double* v1, const double* v2){
		int i = 0;
		for(i = 0; i < N; i++){
			cout << i << endl;
			inp1[i] = v1[i];
			inp2[i] = v2[i];
		}
	}

};

//function declarations -- helper and main
bool main_fcn(bool * call_help1, bool * help_rdy1, double * out_data, help_input_from_main* help_input, bool *interrupt_help);
bool help_fcn(help_input_from_main help_input, double* out);

//function declarations -- calc kernel and monitor kernel
__global__ void dataKernel(int* data, int size);
__global__ void monitorKernel(int* data, int size);


int main()
{
	//define booleans needed for logic
	bool main_done_cmd = 0;
	bool call_help_cmd = 0;	
	bool help_rdy_cmd = 0;
	bool help_running_cmd = 0;
	bool interrupt_help_cmd = 0;

	//define interface between helper and main i.e.: what is returned
	double out_val =0.0;

	//define pointers to bools and doubles
	bool *call_help = &call_help_cmd;
	bool *help_rdy = &help_rdy_cmd;
	bool *main_done = &main_done_cmd;
	bool *help_running = &help_running_cmd;
	bool *interrupt_help = &interrupt_help_cmd;
	double *out = &out_val;

	help_input_from_main* help_input;
	
	cout << "starting set ptr test " << endl;
	const double inp1[N] = {1,2,3,4,5};
	const double inp2[N] = {1,2,3,4,5};
	cout << "WHADDUP I initialized inp1 " << endl;
	(*help_input).initS(&inp1[0], &inp2[0]);	
	cout <<"passed set ptr test " << endl;



	#pragma omp parallel num_threads(2) shared(main_done, help_rdy, call_help, out, help_input, interrupt_help)
	{

	if(omp_get_thread_num() == 0){
		cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
	//code for master threads
		*main_done = main_fcn(call_help, help_rdy, out, help_input, interrupt_help);
	}

	if(omp_get_thread_num() == 1){
		cout <<"WHATDDUP IM LAUNCHING that second" << endl;
		
		while(*main_done == 0){
			//sleep(10);
			//cout << *call_help <<endl;
			//cout << *help_running << endl;
			if(*call_help == 1 && *help_running == 0){
				*help_running = 1;
				*call_help = 0;
				*help_rdy =  help_fcn(*help_input, out);
			}
			//if(*help_running == 1 && *interrupt_help == 0){

		}	
	
	}


	}





	//begin CUDA testing
	const int numElems = 16;
	int hostArray[numElems], *dArray;
	int i = 0;
	hipMalloc((void**)&dArray, sizeof(int)*numElems);
	hipMemset(dArray, 0, numElems*sizeof(int));
	
	dataKernel<<<2, 8>>>(dArray, numElems);
	hipMemcpy(&hostArray, dArray, sizeof(int)*numElems, hipMemcpyDeviceToHost);

	cout << "Values in hostArray: " << endl;
	for(i = 0; i < numElems; i++)
		cout << hostArray[i] << endl;
	hipFree(dArray);


	return 0;

}

bool main_fcn(bool* call_help, bool* help_rdy, double* help_out, help_input_from_main* help_input_ptr, bool *interrupt_help){
	
	cout << "WHADDUP IM IN THE MAIN " << endl;
	//initialize data for input to helper function
	double inp1[N] = {1,2,3,4,5};
	cout << "WHADDUP I initialized inp1 " << endl;
	(*help_input_ptr).initS(inp1, inp1);

	cout << "WHADDUP ABOUT TO CALL DAT HEEELP " << endl;
	//call help function
	*call_help = 1;
	
	cout << "WHADDUP JUST CALLED DAT HELP " << endl;

	//if interrupt not allowed, then sleep until helper function is ready
	if(allow_interrupt == 0){
		while(*help_rdy == 0)
			sleep(1);
	}

	if(allow_interrupt == 1){
		sleep(2); //sleep 2 s to simulate other activities or running code
		*interrupt_help = 1;	//set helper interrupt flag
		while(*help_rdy == 0)    // wait for helper function to finish after interrupt
			sleep(1);
	}

	cout << "helper function returned the following value to main fnc: " << *help_out << endl;
	
	
	
	return 1;


}

bool help_fcn(help_input_from_main help_input, double* out){
	//int j = 1;
	int i = 0;
	double* inp1 = help_input.inp1;
	double* inp2 = help_input.inp2;
	
	for(i = 0; i < N; i++){

		if(i > 0)
			//*out = (*out+inp1[i]+inp2[i])*i/(i+1);
			*out = (*out+inp1[i]+inp2[i]);
		else
			//*out = (*out+inp1[i]+inp2[i]);
			*out = (*out+inp1[i]+inp2[i]);	
		//cout << "out after update = " << *out << endl;	

	
	}
	return 1;
}



__global__ void dataKernel(int* data, int size){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	if(thid < size)
	data[thid] = (blockIdx.x+ threadIdx.x);
}

__global__ void monitorKernel(int* data, int size){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	if(thid < size)
	data[thid] = (blockIdx.x+ threadIdx.x);
}




