#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>

using namespace std;

//global variables
const bool allow_interrupt = 0;
const int N = 5;
__device__ static bool *stop_kernel =0;


struct help_input_from_main{
	static const int length = N;
	double inp1[N];
	double inp2[N];

	void initS(double* v1, double* v2){
		int i = 0;
		for(i = 0; i < N; i++){
			inp1[i] = v1[i];
			inp2[i] = v2[i];
		}
	}

};

struct ctrl_flags{
	bool main_done_cmd = 0;
	bool call_help_cmd = 0;
	volatile bool help_rdy_cmd = 0;
	volatile bool help_running_cmd = 0;
	volatile bool interrupt_help_cmd = 0;
	volatile bool request_val_cmd = 0;
	volatile bool req_delivered_cmd = 0;
	
	bool *call_help = &call_help_cmd;
	volatile bool *help_rdy = &help_rdy_cmd;
	bool *main_done = &main_done_cmd;
	volatile bool *help_running = &help_running_cmd;
	volatile bool *interrupt_help = &interrupt_help_cmd;
	volatile bool *request_val = &request_val_cmd;
	volatile bool *request_done = &req_delivered_cmd;};

//function declarations -- helper and main
bool main_fcn(ctrl_flags CF, int * out_data, help_input_from_main* help_input);
bool help_fcn(help_input_from_main help_input, double* out);
bool init_help(help_input_from_main help_input);

//function declarations -- calc kernel and monitor kernel
__global__ void dataKernel( int* data,  bool *stop_kern_ptr);
__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr);


int main()
{
	//define booleans needed for logic
	ctrl_flags CF;

	//define interface between helper and main i.e.: what is returned
	int out_val =0.0;


	int *out = &out_val;

	help_input_from_main test_input;	
	help_input_from_main* help_input = &test_input;

	static double inp1[N] = {1,2,3,4,5};
	static double inp2[N] = {1,2,3,4,5};

	(*help_input).initS(&inp1[0], &inp2[0]);	


	#pragma omp parallel num_threads(2) shared(CF)
	{

		if(omp_get_thread_num() == 0){
			cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
		//code for master threads
			CF.main_done_cmd = main_fcn(CF, out, help_input);
		}

		if(omp_get_thread_num() == 1){
			cout <<"Running CUDA init" << endl;
			const int numElems = 1;
			int hostArray[numElems];
			int *dArray;

			int i = 0;

			//pointer of helper function return	
			int transfered_data;
			int *h_data = &transfered_data;
			int *monitor_data;

		

			bool *host_stop_kernel;
		
			hipMalloc(&stop_kernel, sizeof(bool));
			hipHostMalloc((void**)&host_stop_kernel, sizeof(bool), hipHostMallocDefault);
			*host_stop_kernel = 0;			
			bool *stop_kern_ptr;
			hipGetSymbolAddress((void**)&stop_kern_ptr, HIP_SYMBOL(stop_kernel));

			hipMalloc((void**)&dArray, sizeof(int)*numElems);
			//cudaMalloc((void**)&dArray_Held, sizeof(int)*numElems);
			hipMemset(dArray, 0, numElems*sizeof(int));
			//cudaMemset(dArray_Held, 0, numElems*sizeof(int));
			hipHostMalloc((void**)&monitor_data, sizeof(int), hipHostMallocDefault);
			hipStream_t stream1;
			hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);


			while(CF.main_done_cmd == 0){

				if(CF.call_help_cmd == 1 && CF.help_running_cmd == 0){
					CF.help_running_cmd = 1;
					CF.call_help_cmd = 0;
					cout <<"Launching Helper Kernel" << endl;
					//*help_rdy =  help_fcn(*help_input, out);
					dataKernel<<<1, 4>>>(dArray, stop_kern_ptr);
				}
				if(CF.help_running_cmd == 1 && allow_interrupt == 0 && CF.request_val_cmd == 1){	
					cout <<"Launching Monitor Kernel" << endl;
					//cudaStreamSynchronize(stream1);
					monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dArray);
					cout <<"Launching Async Mem Cpy" << endl;
					hipMemcpyAsync(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost, stream1);
					hipStreamSynchronize(stream1);
					CF.request_val_cmd = 0;
					*out = *h_data;
					CF.req_delivered_cmd = 1;
				}	
			}

			*host_stop_kernel = 1;

			cout <<"Trying to Stop Helper Kernel" << endl;
			hipMemcpyAsync(stop_kern_ptr, host_stop_kernel, sizeof(bool), hipMemcpyHostToDevice, stream1);
			hipStreamSynchronize(stream1);

			//cout << "Copying values from helper kernel to base (but they may be garbage!!!!!" << endl;
			//cudaMemcpy(&hostArray, dArray_Held, sizeof(int)*numElems, cudaMemcpyDeviceToHost);


			for(i = 0; i < numElems; i++)
				cout << hostArray[i] << endl;

			hipFree(dArray);
			hipFree(monitor_data);
			cout << "Expected h_data point to: " << *h_data << endl;
	
		}



	}


	return 0;

}

bool main_fcn(ctrl_flags CF, int* help_out, help_input_from_main* help_input_ptr)
{	
	bool *call_help = CF.call_help;
	//volatile bool *help_rdy = CF.help_rdy;
	volatile bool *request_val = CF.request_val;
	volatile bool *request_done = CF.request_done;

	//initialize data for input to helper function
	double inp1[N] = {1,2,3,4,5};
	

	//set values of helper function input
	(*help_input_ptr).initS(inp1, inp1);
	//ask to start help function	
	cout << "Main calling help function for 1st time" << endl;
	*call_help = 1;
	
	//some code/processing goes here
	sleep(1);

	//if interrupt not allowed, then request value from help
	if(allow_interrupt == 0){	
		cout << "Main requesting function update" << endl;
		*request_val = 1;
		while(*request_done == 0)
			sleep(1);
	}
	//..cout << "Main requesting function update" << endl;

	/*if(allow_interrupt == 1){
		sleep(2); //sleep 2 s to simulate other activities or running code
		*interrupt_help = 1;	//set helper interrupt flag
		while(*help_rdy == 0)    // wait for helper function to finish after interrupt
			sleep(1);
	}*/

	cout << "Main update received " << *help_out << endl;
	*request_done = 0;
	sleep(2);

	cout << "Main Requestiong Second function update " << endl;
	cout << "Current Request Val (shoudl be 0) = " << *request_val << endl;
	
	*request_val = 1;
	while(*request_done == 0)
		sleep(1);
	cout << "Main update received " << *help_out << endl;
	*request_done = 0;
	sleep(2);

	cout << "Main Requestiong Third function update " << endl;
	*request_val = 1;
	while(*request_done == 0)
		sleep(1);
	cout << "Main update received " << *help_out << endl;
	*request_done = 0;
	//sleep(2);
	cout << "Exiting Main" << endl;
	
	return 1;


}

bool help_fcn(help_input_from_main help_input, double* out){
	//int j = 1;
	int i = 0;
	double* inp1 = help_input.inp1;
	double* inp2 = help_input.inp2;
	
	for(i = 0; i < N; i++){

		if(i > 0)
			//*out = (*out+inp1[i]+inp2[i])*i/(i+1);
			*out = (*out+inp1[i]+inp2[i]);
		else
			//*out = (*out+inp1[i]+inp2[i]);
			*out = (*out+inp1[i]+inp2[i]);	
		//cout << "out after update = " << *out << endl;	

	
	}
	return 1;
}

bool init_help(help_input_from_main help_input){
	

return 1;

}



__global__ void dataKernel( int* data, bool *stop){
//this adds a value to a variable stored in global memory
	*data = 3;

	while(1){
		if(*data > 300)
			*data = 0;
		*data = *data+1;
		if(*stop == 1){
			*data = 6;
			__syncthreads();
			asm("trap;");
		}

		
	}	





/* int thid = threadIdx.x+blockIdx.x*blockDim.x;

	if(thid < size){
		data_held[thid] = (blockIdx.x+ threadIdx.x);
		data[thid] = (blockIdx.x+ threadIdx.x);
		while(1){
			if(data[thid] < 1000)
				data[thid] = data[thid]+.2;
			else
				data[thid] = data[thid]-100;
			if(*stop_kern_ptr == 1){
					__syncthreads();

					asm("trap;");
					}
			


		}
	}*/

}


__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr){

	*write_2_ptr = *read_in_ptr;


}

