#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>
#include <sys/time.h>
#include "test_kernels.h"
#include "main_fcn.h"

using namespace std;



int main()
{
	//define booleans needed for logic
	ctrl_flags CF;


	//define interface between helper and main i.e.: what is returned
	//double out_val =0.0;


	double out[numElems];
	double temp_out[numElems];

	help_input_from_main test_input;	
	help_input_from_main* help_input = &test_input;

	static double inp1[N] = {5};


	(*help_input).initS(&inp1[0]);	

			int i = 0;
	
			//pointer of helper function return	



	#pragma omp parallel num_threads(3) shared(CF, help_input, out)
	{

		if(omp_get_thread_num() == 0){
			cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
		//code for master threads
			CF.main_done_cmd = main_fcn(CF, temp_out, help_input);
		}

		if(omp_get_thread_num() == 1){

			//hipStreamCreate(&stream1);


			while(CF.main_done_cmd == 0){

				if(CF.call_help_cmd == 1 && CF.help_running_cmd == 0){
					CF.help_running_cmd = 1;
					CF.call_help_cmd = 0;
					cout <<"Launching Helper Kernel" << endl;
					//*help_rdy =  help_fcn(*help_input, out);
					//sleep(10);
					CF.help_rdy_cmd = help_fcn(*help_input, out, &CF.kernel_rdy_cmd);
				}
				
			}


		
			for(i = 0; i < 5; i++)
 				cout << "Last value of helper function: "  << out[i] << endl;

	
		}
		if(omp_get_thread_num() == 2){
			while(CF.main_done_cmd == 0){
				if(CF.help_running_cmd == 1 && allow_interrupt == 0 && CF.request_val_cmd == 1){	
					cout <<"Launching Monitor" << endl;
	
					CF.request_val_cmd = 0;
				
					
					for(i = 0; i < numElems; i++){
						temp_out[i] = out[i];
						if(i < 3)
							cout << "value monitored over: " << temp_out[i] << endl;

					}
					CF.req_delivered_cmd = 1;
				}	
			
			}

		}



	}


	return 0;

}



