#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>

using namespace std;

//global variables
const bool allow_interrupt = 0;
const int N = 5;
__device__ bool *stop_kernel =0;

struct help_input_from_main{
	static const int length = N;
	double inp1[N];
	double inp2[N];

	void initS(double* v1, double* v2){
		int i = 0;
		for(i = 0; i < N; i++){
			inp1[i] = v1[i];
			inp2[i] = v2[i];
		}
	}

};

struct ctrl_flags{
	bool main_done_cmd = 0;
	bool call_help_cmd = 0;
	bool help_rdy_cmd = 0;
	bool help_running_cmd = 0;
	bool interrupt_help_cmd = 0;
	bool request_val_cmd = 0;
	bool req_delivered_cmd = 0;
	
	bool *call_help = &call_help_cmd;
	bool *help_rdy = &help_rdy_cmd;
	bool *main_done = &main_done_cmd;
	bool *help_running = &help_running_cmd;
	bool *interrupt_help = &interrupt_help_cmd;
	bool *request_val = &request_val_cmd;
	bool *request_done = &req_delivered_cmd;};

//function declarations -- helper and main
bool main_fcn(ctrl_flags CF, double * out_data, help_input_from_main* help_input);
bool help_fcn(help_input_from_main help_input, double* out);
bool init_help(help_input_from_main help_input);

//function declarations -- calc kernel and monitor kernel
__global__ void dataKernel( int* data, int size);
__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr);


int main()
{
	//define booleans needed for logic
	ctrl_flags CF;

	//define interface between helper and main i.e.: what is returned
	double out_val =0.0;


	double *out = &out_val;

	help_input_from_main test_input;	
	help_input_from_main* help_input = &test_input;

	static double inp1[N] = {1,2,3,4,5};
	static double inp2[N] = {1,2,3,4,5};

	(*help_input).initS(&inp1[0], &inp2[0]);	


	#pragma omp parallel num_threads(2) shared(CF)
	{

		if(omp_get_thread_num() == 0){
			cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
		//code for master threads
			CF.main_done_cmd = main_fcn(CF, out, help_input);
		}

		if(omp_get_thread_num() == 1){
			cout <<"Running CUDA init" << endl;
			const int numElems = 4;
			int hostArray[numElems];
			int *dArray;
			int i = 0;

			//pointer of helper function return	
			int transfered_data;
			int *h_data = &transfered_data;
			int *monitor_data;

			//bool *stop_kern_ptr = &stop_kernel;
		
			hipMalloc(&stop_kernel, sizeof(bool));
			hipMalloc((void**)&dArray, sizeof(int)*numElems);
			hipMemset(dArray, 0, numElems*sizeof(int));
			hipMalloc((void**)&monitor_data, sizeof(int));
			hipMalloc((void**)&dArray, sizeof(int)*numElems);

			hipStream_t stream1;
			hipStreamCreate(&stream1);


			while(CF.main_done_cmd == 0){

				if(CF.call_help_cmd == 1 && CF.help_running_cmd == 0){
					CF.help_running_cmd = 1;
					CF.call_help_cmd = 0;
					cout <<"Launching Helper Kernel" << endl;
					//*help_rdy =  help_fcn(*help_input, out);
					dataKernel<<<1, 4>>>(dArray, numElems);
				}
				if(CF.help_running_cmd == 1 && allow_interrupt == 0 && CF.request_val_cmd == 1){	
					cout <<"Launching Monitor Kernel" << endl;
					hipDeviceSynchronize(stream1);
					monitorKernel<<<1, 1,0, stream1>>>(monitor_data, &dArray[2]);
					cout <<"Launching Async Mem Cpy" << endl;
					hipMemcpyAsync(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost, stream1);
					hipDeviceSynchronize(stream1);
					CF.request_val_cmd = 0;
					*out = *h_data;
					CF.req_delivered_cmd = 1;
				}	
			}

			bool stop_kernel = 1;
			bool *host_stop_kernel = &stop_kernel;
			cout <<"Trying to Stop Helper Kernel" << endl;
			hipMemcpyAsync(&stop_kernel, host_stop_kernel, sizeof(bool), hipMemcpyHostToDevice, stream1);

			cout << "Copying values from helper kernel to base (but they may be garbage!!!!!" << endl;
			hipMemcpy(&hostArray, dArray, sizeof(int)*numElems, hipMemcpyDeviceToHost);


			for(i = 0; i < numElems; i++)
				cout << hostArray[i] << endl;

			hipFree(dArray);
			hipFree(monitor_data);
			cout << "Expected h_data point to: " << *h_data << endl;
	
		}



	}


	return 0;

}

bool main_fcn(ctrl_flags CF, double* help_out, help_input_from_main* help_input_ptr)
{	
	bool *call_help = CF.call_help;
	bool *help_rdy = CF.help_rdy;
	bool *request_val = CF.request_val;
	bool *request_done = CF.request_done;

	//initialize data for input to helper function
	double inp1[N] = {1,2,3,4,5};
	

	//set values of helper function input
	(*help_input_ptr).initS(inp1, inp1);
	//ask to start help function	
	cout << "Main calling help function for 1st time" << endl;
	*call_help = 1;
	
	//some code/processing goes here
	sleep(1);

	//if interrupt not allowed, then request value from help
	if(allow_interrupt == 0){	
		cout << "Main requesting function update" << endl;
		*request_val = 1;
		while(*request_done == 0)
			sleep(1);
	}
	//..cout << "Main requesting function update" << endl;

	/*if(allow_interrupt == 1){
		sleep(2); //sleep 2 s to simulate other activities or running code
		*interrupt_help = 1;	//set helper interrupt flag
		while(*help_rdy == 0)    // wait for helper function to finish after interrupt
			sleep(1);
	}*/

	cout << "Main update received " << *help_out << endl;
	*request_done = 0;
	sleep(2);

	cout << "Main Requestiong Second function update " << endl;
	cout << "Current Request Val (shoudl be 0) = " << *request_val << endl;
	
	*request_val = 1;
	while(*request_done == 0)
		sleep(1);
	cout << "Main update received " << *help_out << endl;
	*request_done = 0;
	sleep(2);

	cout << "Main Requestiong Third function update " << endl;
	*request_val = 1;
	while(*request_done == 0)
		sleep(1);
	cout << "Main update received " << *help_out << endl;
	*request_done = 0;
	//sleep(2);
	cout << "Exiting Main" << endl;
	
	return 1;


}

bool help_fcn(help_input_from_main help_input, double* out){
	//int j = 1;
	int i = 0;
	double* inp1 = help_input.inp1;
	double* inp2 = help_input.inp2;
	
	for(i = 0; i < N; i++){

		if(i > 0)
			//*out = (*out+inp1[i]+inp2[i])*i/(i+1);
			*out = (*out+inp1[i]+inp2[i]);
		else
			//*out = (*out+inp1[i]+inp2[i]);
			*out = (*out+inp1[i]+inp2[i]);	
		//cout << "out after update = " << *out << endl;	

	
	}
	return 1;
}

bool init_help(help_input_from_main help_input){
	

return 1;

}



__global__ void dataKernel( int* data, int size){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	int i = 0;
	if(thid < size){
		data[thid] = (blockIdx.x+ threadIdx.x);
		while(1){
			if(data[thid] < 50000)
				data[thid] = data[thid]+.2;
			else
				data[thid] = data[thid]-10000;
			if(*stop_kernel == 1){
					__threadfence();
					asm("trap;");
					}
					
	


		}
	}

}


__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr){
	*write_2_ptr = *read_in_ptr;

}

