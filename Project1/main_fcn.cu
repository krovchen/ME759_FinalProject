#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>
#include <sys/time.h>
#include "test_kernels.h"
#include "main_fcn.h"

using namespace std;



int main()
{
	//define booleans needed for logic
	ctrl_flags CF;


	//define interface between helper and main i.e.: what is returned
	//double out_val =0.0;


	double out[numElems];

	help_input_from_main test_input;	
	help_input_from_main* help_input = &test_input;

	static double inp1[N] = {5};


	(*help_input).initS(&inp1[0]);	


	#pragma omp parallel num_threads(2) shared(CF, help_input, out)
	{

		if(omp_get_thread_num() == 0){
			cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
		//code for master threads
			CF.main_done_cmd = main_fcn(CF, out, help_input);
		}

		if(omp_get_thread_num() == 1){
			cout <<"Running CUDA init" << endl;

			//EDIT THIS to init cuda
			double *dArray;

			int i = 0;
			int numBlocks = 1;
			int numThreads = numElems;
			if(numElems > 1024){			//for now just assume numElems is multiple of 1024
				numThreads = 1024;
				numBlocks = numElems/numThreads;
			}

			//pointer of helper function return	

			double* h_data;
			double* monitor_data;
			

			

			hipMalloc((void**)&dArray, sizeof(double)*numElems);
			hipMemset(dArray, 0, numElems*sizeof(double));
		
			hipMalloc((void**)&monitor_data, sizeof(double)*numElems);
			hipHostMalloc((void**)&h_data, sizeof(double)*numElems);
			hipStream_t stream1;
			hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
		


			while(CF.main_done_cmd == 0){

				if(CF.call_help_cmd == 1 && CF.help_running_cmd == 0){
					CF.help_running_cmd = 1;
					CF.call_help_cmd = 0;
					cout <<"Launching Helper Kernel" << endl;
					//*help_rdy =  help_fcn(*help_input, out);
			
					dataKernel<<<numBlocks,numThreads>>>(dArray, 1000);
				}
				if(CF.help_running_cmd == 1 && allow_interrupt == 0 && CF.request_val_cmd == 1){	
					cout <<"Launching Monitor Kernel" << endl;
					//hipStreamSynchronize(stream1);
					monitorKernel<<<numBlocks, numThreads,0, stream1>>>(monitor_data, dArray);
					cout <<"Launching Async Mem Cpy" << endl;
					hipMemcpyAsync(h_data, monitor_data, numElems*sizeof(double), hipMemcpyDeviceToHost, stream1);
					hipStreamSynchronize(stream1);
					CF.request_val_cmd = 0;
					for(i = 0; i < numElems; i++)
						out[i] = h_data[i];
					CF.req_delivered_cmd = 1;
				}	
			}


			hipMemcpy(h_data, dArray, sizeof(double)*numElems, hipMemcpyDeviceToHost);
			for(i = 0; i < 5; i++)
				cout << "Value copied over: "  << h_data[i] << endl;

			hipFree(dArray);
			hipFree(monitor_data);
		
	
		}



	}


	return 0;

}



