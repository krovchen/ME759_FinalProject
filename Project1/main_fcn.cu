#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>

using namespace std;

//global variables
const bool allow_interrupt = 0;
const int N = 5;
bool stop_kernel =0;

struct help_input_from_main{
	static const int length = N;
	double inp1[N];
	double inp2[N];

	void initS(double* v1, double* v2){
		int i = 0;
		for(i = 0; i < N; i++){
			inp1[i] = v1[i];
			inp2[i] = v2[i];
		}
	}

};

//function declarations -- helper and main
bool main_fcn(bool * call_help1, bool * help_rdy1, double * out_data, help_input_from_main* help_input, bool *interrupt_help);
bool help_fcn(help_input_from_main help_input, double* out);

//function declarations -- calc kernel and monitor kernel
__global__ void dataKernel( int* data, int size);
__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr);


int main()
{
	//define booleans needed for logic
	bool main_done_cmd = 0;
	bool call_help_cmd = 0;	
	bool help_rdy_cmd = 0;
	bool help_running_cmd = 0;
	bool interrupt_help_cmd = 0;

	//define interface between helper and main i.e.: what is returned
	double out_val =0.0;

	//define pointers to bools and doubles
	bool *call_help = &call_help_cmd;
	bool *help_rdy = &help_rdy_cmd;
	bool *main_done = &main_done_cmd;
	bool *help_running = &help_running_cmd;
	bool *interrupt_help = &interrupt_help_cmd;
	double *out = &out_val;


	
	help_input_from_main test_input;
	
	help_input_from_main* help_input = &test_input;


	static double inp1[N] = {1,2,3,4,5};
	static double inp2[N] = {1,2,3,4,5};

	(*help_input).initS(&inp1[0], &inp2[0]);	




	#pragma omp parallel num_threads(2) shared(main_done, help_rdy, call_help, out, help_input, interrupt_help)
	{

	if(omp_get_thread_num() == 0){
		cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
	//code for master threads
		*main_done = main_fcn(call_help, help_rdy, out, help_input, interrupt_help);
	}

	if(omp_get_thread_num() == 1){
		cout <<"WHATDDUP IM LAUNCHING that second" << endl;
		
		while(*main_done == 0){
			//sleep(10);
			//cout << *call_help <<endl;
			//cout << *help_running << endl;
			if(*call_help == 1 && *help_running == 0){
				*help_running = 1;
				*call_help = 0;
				*help_rdy =  help_fcn(*help_input, out);
			}
			//if(*help_running == 1 && *interrupt_help == 0){

		}	
	
	}


	}




	cout << "begin CUDA Testing" << endl;
	//begin CUDA testing
	const int numElems = 4;
	int hostArray[numElems];
	 int *dArray;
	int i = 0;

	//pointer of helper function return	
	int transfered_data;
	 int *h_data = &transfered_data;
	int *monitor_data;

	bool *stop_kern_ptr = &stop_kernel;
	//cudaSetDeviceFlags(cudaDeviceMapHost);
	//cudaHostAlloc((void **)&h_data, sizeof(int), cudaHostAllocMapped);
	//cudaHostGetDevicePointer((int **)&d_data, (int *)h_data,0);
	
	hipMalloc(&stop_kern_ptr, sizeof(bool));
	hipMalloc((void**)&dArray, sizeof(int)*numElems);
	hipMemset(dArray, 0, numElems*sizeof(int));
	hipMalloc((void**)&monitor_data, sizeof(int));
	hipMalloc((void**)&dArray, sizeof(int)*numElems);



	dataKernel<<<1, 4>>>(dArray, numElems);

	monitorKernel<<<1, 1>>>(monitor_data, &dArray[2]);

	hipMemcpy(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy(&hostArray, dArray, sizeof(int)*numElems, hipMemcpyDeviceToHost);


	for(i = 0; i < numElems; i++)
		cout << hostArray[i] << endl;

	hipFree(dArray);

	hipFree(monitor_data);



	cout << "Expected h_data to point to 1, actual point to: " << *h_data << endl;

	return 0;


}

bool main_fcn(bool* call_help, bool* help_rdy, double* help_out, help_input_from_main* help_input_ptr, bool *interrupt_help){
	
	cout << "WHADDUP IM IN THE MAIN " << endl;
	//initialize data for input to helper function
	double inp1[N] = {1,2,3,4,5};
	cout << "WHADDUP I initialized inp1 " << endl;
	(*help_input_ptr).initS(inp1, inp1);

	cout << "WHADDUP ABOUT TO CALL DAT HEEELP " << endl;
	//call help function
	*call_help = 1;
	
	cout << "WHADDUP JUST CALLED DAT HELP " << endl;

	//if interrupt not allowed, then sleep until helper function is ready
	if(allow_interrupt == 0){
		while(*help_rdy == 0)
			sleep(1);
	}

	if(allow_interrupt == 1){
		sleep(2); //sleep 2 s to simulate other activities or running code
		*interrupt_help = 1;	//set helper interrupt flag
		while(*help_rdy == 0)    // wait for helper function to finish after interrupt
			sleep(1);
	}

	cout << "helper function returned the following value to main fnc: " << *help_out << endl;
	
	
	
	return 1;


}

bool help_fcn(help_input_from_main help_input, double* out){
	//int j = 1;
	int i = 0;
	double* inp1 = help_input.inp1;
	double* inp2 = help_input.inp2;
	
	for(i = 0; i < N; i++){

		if(i > 0)
			//*out = (*out+inp1[i]+inp2[i])*i/(i+1);
			*out = (*out+inp1[i]+inp2[i]);
		else
			//*out = (*out+inp1[i]+inp2[i]);
			*out = (*out+inp1[i]+inp2[i]);	
		//cout << "out after update = " << *out << endl;	

	
	}
	return 1;
}



__global__ void dataKernel( int* data, int size){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	if(thid < size)
	data[thid] = (blockIdx.x+ threadIdx.x);

}


__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr){
//this adds a value to a variable stored in global memory
	*write_2_ptr = *read_in_ptr;

}

