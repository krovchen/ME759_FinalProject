#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>
#include <sys/time.h>
#include "test_kernels.h"
#include "main_fcn.h"
#include <stdlib.h>

using namespace std;

void gen_b_vec(double* inp1){
	int i = 0;
	for(i = 0; i < Ni; i++)
		inp1[i] = i*.01;
}

void gen_A_mat(double* A)
//generate A matrix that is diagonally dominant
{
	int i = 0;
	int d = 0;
	int temp;
	double val;
	for(i = 0; i < NumElems; i++){
		temp = rand();
		A[i] = (double)temp/RAND_MAX;
		if(i == d*Ni+d){			//this part makes the matrix diagonally dominant
			A[i] = A[i]+Ni;
			d=d+1;
		
		}

	}
}


int main()
{
	//define booleans needed for logic
	ctrl_flags CF;


	//define interface between helper and main i.e.: what is returned
	//double out_val =0.0;


	double out[Ni];
	double Amat[numElems];

	gen_A_mat(Amat);

	help_input_from_main test_input;	
	help_input_from_main* help_input = &test_input;

	static double inp1[Ni];
	gen_b_vec(inp1);

	(*help_input).initS(&inp1[0]);	
		cout <<"Running CUDA init" << endl;

			double *x_now_d, *x_next_d, *A_d, *b_d;
			int k;

			//pointer of helper function return	

			double* h_data;
			double* monitor_data;
			
			    // Allocate memory on the device
			hipMalloc((void **) &x_next_d, Ni*sizeof(double)));
			hipMalloc((void **) &A_d, NumElems*sizeof(double)));
 			hipMalloc((void **) &x_now_d, Ni*sizeof(double)));
	 		hipMalloc((void **) &b_d, Ni*sizeof(double)));
	
			hipMalloc((void**)&monitor_data, sizeof(double)*Ni);
			hipHostMalloc((void**)&h_data, sizeof(double)*Ni);
			hipStream_t stream1;
			hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
		
			 // Optimized kernel
   			 int nTiles = Ni/tileSize + (Ni%tileSize == 0?0:1);
   			 int gridHeight = Nj/tileSize + (Nj%tileSize == 0?0:1);
   			 int gridWidth = Ni/tileSize + (Ni%tileSize == 0?0:1);
    		
    		dim3 dGrid(gridHeight, gridWidth), dBlock(tileSize, tileSize);

	#pragma omp parallel num_threads(3) shared(CF, help_input, out)
	{

		if(omp_get_thread_num() == 0){
			cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
		//code for master threads
			CF.main_done_cmd = main_fcn(CF, out, help_input);
		}

		if(omp_get_thread_num() == 1){
	

			while(CF.main_done_cmd == 0){


				if(CF.help_running_cmd == 1 && allow_interrupt == 0 && CF.request_val_cmd == 1){	
					cout <<"Launching Monitor Kernel" << endl;
					//hipStreamSynchronize(stream1);
					monitorKernel<<<numBlocks, numThreads,0, stream1>>>(monitor_data, x_now_d);
					cout <<"Launching Async Mem Cpy" << endl;
					hipMemcpyAsync(h_data, monitor_data, numElems*sizeof(double), hipMemcpyDeviceToHost, stream1);
					hipStreamSynchronize(stream1);
					CF.request_val_cmd = 0;
					for(i = 0; i < numElems; i++)
						out[i] = h_data[i];
					CF.req_delivered_cmd = 1;
				}	
			}



		
	
		}
		if(omp_get_thread_num() == 2){
			while(CF.main_done_cmd == 0){
				if(CF.call_help_cmd == 1 && CF.help_running_cmd == 0){

					cout <<"Launching Helper Function" << endl;
					//*help_rdy =  help_fcn(*help_input, out);
					CF.help_running_cmd = 1;
					CF.call_help_cmd = 0;
					dataKernel<<<nTiles, tileSize >>>(dArray, 1000);

				}
			}
			hipMemcpy(h_data, dArray, sizeof(double)*numElems, 				hipMemcpyDeviceToHost);
			for(i = 0; i < 5; i++)
				cout << "Value copied over: "  << h_data[i] << endl;

			hipFree(dArray);
			hipFree(monitor_data);
	
		}


	}


	return 0;

}



