#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>
#include <sys/time.h>
#include "jacobi_kernels.h"
#include "main_fcn.h"
#include <stdlib.h>

using namespace std;




int main()
{
	//define booleans needed for logic
	ctrl_flags CF;


	//define interface between helper and main i.e.: what is returned
	//double out_val =0.0;

	int i;
	double out[Ni];
	double Amat[numElems];
	int numBlocks, numThreads;

	gen_A_mat(Amat);

	static double inp1[Ni];
	gen_b_vec(inp1);

	help_input_from_main test_input;	
	test_input.initS(inp1, Amat);	
	

	cout <<"Running CUDA init" << endl;

	double *x_now_d, *x_next_d, *A_d, *b_d;
	int k;

	//pointer of helper function return
	double* h_data;
	double* monitor_data;  //asdfasdf
			
	    // Allocate memory on the device
	hipMalloc((void **) &x_next_d, Ni*sizeof(double));
	hipMalloc((void **) &A_d, numElems*sizeof(double));
 	hipMalloc((void **) &x_now_d, Ni*sizeof(double));
	hipMalloc((void **) &b_d, Ni*sizeof(double));
	
	hipMalloc((void**)&monitor_data, sizeof(double)*Ni);
	hipHostMalloc((void**)&h_data, sizeof(double)*Ni);

	test_input.x_next_d = x_next_d;
	test_input.A_d = A_d;
	test_input.b_d = b_d;
	test_input.x_now_d = x_now_d;

	test_input.nTiles = Ni/tileSize + (Ni%tileSize == 0?0:1);

	help_input_from_main* help_input = &test_input;

	hipStream_t stream1;
	hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
	
	numBlocks = 1;
	numThreads = els_to_read;
	if(els_to_read > 1024){			//for now just assume numElems is multiple of 1024 or less
		numThreads = 1024;
		numBlocks = els_to_read/numThreads;
	}
	 // Optimized kernel
   	 
   	 //int gridHeight = Nj/tileSize + (Nj%tileSize == 0?0:1);
   	// int gridWidth = Ni/tileSize + (Ni%tileSize == 0?0:1);
    		
    	//dim3 dGrid(gridHeight, gridWidth), dBlock(tileSize, tileSize);

	#pragma omp parallel num_threads(3) shared(CF, help_input, out)
	{

		if(omp_get_thread_num() == 0){
			cout <<"WHATDDUP IM LAUNCHING THAT MAIN" << endl;
		//code for master threads
			CF.main_done_cmd = main_fcn(CF, out, help_input);
		}

		if(omp_get_thread_num() == 1){
			while(CF.main_done_cmd == 0){
				if(CF.help_running_cmd == 1 && CF.request_val_cmd == 1){	
					cout <<"Launching Monitor Kernel" << endl;
					//hipStreamSynchronize(stream1);
					monitorKernel<<<numBlocks, numThreads,0, stream1>>>(monitor_data, test_input.x_now_d);
					cout <<"Launching Async Mem Cpy" << endl;
					hipMemcpyAsync(h_data, monitor_data, Ni*sizeof(double), hipMemcpyDeviceToHost, stream1);
					hipStreamSynchronize(stream1);
					CF.request_val_cmd = 0;
					for(i = 0; i < Ni; i++){
						out[i] = h_data[i];
						if(i < 3)
						cout << "value copied in monitor kernel: " << h_data[i] << endl;
					}
					CF.req_delivered_cmd = 1;
				}	
			}	
		}
		if(omp_get_thread_num() == 2){
			while(CF.main_done_cmd == 0){
				if(CF.call_help_cmd == 1 && CF.help_running_cmd == 0){

					cout <<"Launching Helper Function" << endl;
					//*help_rdy =  help_fcn(*help_input, out);
					CF.help_running_cmd = 1;
					CF.call_help_cmd = 0;
					CF.help_rdy_cmd = help_fcn(*help_input, out);
					//dataKernel<<<nTiles, tileSize >>>(dArray, 1000);

				}
			}
			
	
		}
		

	}

	
	hipFree(x_next_d);
	hipFree(A_d);
	hipFree(x_now_d);
	hipFree(b_d);
	hipFree(monitor_data);
	hipFree(h_data);
	
	return 0;

}



