#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>
#include <sys/time.h>
#include "jacobi_kernels.h"
#include "main_fcn.h"


using namespace std;

void gen_b_vec(double* inp1){
	int i = 0;
	for(i = 0; i < Ni; i++)
		inp1[i] = i*.01;
}

void gen_A_mat(double* A)
//generate A matrix that is diagonally dominant
{
	int i = 0;
	int d = 0;
	int temp;
	double val;
	for(i = 0; i < numElems; i++){
		temp = rand();
		A[i] = (double)temp/RAND_MAX;
		if(i == d*Ni+d){			//this part makes the matrix diagonally dominant
			A[i] = A[i]+Ni;
			d=d+1;
		
		}

	}
}


__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr){

	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	write_2_ptr[thid] = read_in_ptr[thid];


}

__global__ void jacobiOptimizedOnDevice(double* x_next, double* A, double* x_now, double* b, int Ni, int Nj)
{
   // Optimization step 1: tiling
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
     
    if (idx < Ni)
    {
        double sigma = 0.0;

        // Optimization step 2: store index in register
        // Multiplication is not executed in every iteration.
        int idx_Ai = idx*Nj;
        
        // Tried to use prefetching, but then the result is terribly wrong and I don't know why.. 
        /*     
        float curr_A = A[idx_Ai];
        float nxt_A;
        //printf("idx=%d\n",idx);
        for (int j=0; j<Nj-1; j++)
        {
            if (idx != j)
                nxt_A = A[idx_Ai + j + 1];
                sigma += curr_A * x_now[j];
                //sigma += A[idx_Ai + j] * x_now[j];
                curr_A = nxt_A;
                //printf("curr_A=%f\n",curr_A);
        }
        if (idx != Nj-1)
            sigma += nxt_A * x_now[Nj-1];
        x_next[idx] = (b[idx] - sigma) / A[idx_Ai + idx];
        */
        
        for (int j=0; j<Nj; j++)
            if (idx != j)
                sigma += A[idx_Ai + j] * x_now[j];

        // Tried to use loop-ennrolling, but also here this gives a wrong result.. 
        /*
        for (int j=0; j<Nj/4; j+=4)
        {
            if (idx != j)
            {
                sigma += A[idx_Ai + j] * x_now[j];
            }
            if (idx != j+1)
            {
                sigma += A[idx_Ai + j+1] * x_now[j+1];
            }
            if (idx != j+2)
            {
               sigma += A[idx_Ai + j+2] * x_now[j+2];
            }
            if (idx != j+3)
            {
                sigma += A[idx_Ai + j+3] * x_now[j+3];
            }
        }*/

        x_next[idx] = (b[idx] - sigma) / A[idx_Ai + idx];
    }
    
}




bool help_fcn(help_input_from_main help_input, double* out){
	//int j = 1;
	int k = 0;
	double* x_now_d;
	x_now_d = help_input.x_now_d;
	double* A_d = help_input.A_d;
	double *x_next_d = help_input.x_next_d;
	double *b_d = help_input.b_d;
	double nTiles = help_input.nTiles;
	double *b_h = help_input.b_h;
	double *A_h = help_input.A_h;	

	hipMemcpy(b_d, b_h, sizeof(double)*Ni, hipMemcpyHostToDevice);
	hipMemcpy(A_d, A_h, sizeof(double)*numElems, hipMemcpyHostToDevice);
	for(k = 0; k < 5; k++){
		cout << "copied from A: " << A_h[k] << endl;
		cout << "copied from b: " << b_h[k] << endl;

	}

        for (k=0; k<iter; k++)
        {
            if (k%2)
                jacobiOptimizedOnDevice <<< nTiles, tileSize >>> (x_now_d, A_d, x_next_d, b_d, Ni, Nj);
            else
                jacobiOptimizedOnDevice <<< nTiles, tileSize >>> (x_next_d, A_d, x_now_d, b_d, Ni, Nj);
            //hipMemcpy(x_now_d, x_next_d, sizeof(float)*Ni, hipMemcpyDeviceToDevice);

		sleep(.01);
        }
	
	hipMemcpy(out, x_now_d, sizeof(double)*Ni, hipMemcpyDeviceToHost);
	cout << "finished copy" << endl;
	for(k = 0; k < 5; k++)
		cout << "Value copied over: "  << out[k] << endl;
	cout << "exiting help" << endl;

	return 1;
}



bool main_fcn(ctrl_flags CF, double* help_out, help_input_from_main* help_input_ptr)
{	
	struct timeval stop, start;
	bool *call_help = CF.call_help;
	//volatile bool *help_rdy = CF.help_rdy;
	volatile bool *request_val = CF.request_val;
	volatile bool *request_done = CF.request_done;

	//initialize data for input to helper function
	//double inp1[N] = {4};
	int i = 0;
	int numReads = 10;
	double sval;
	double sum_times = 0;
	int j = 0;

	double Amat[numElems];
	double b_h[Ni];
	gen_A_mat(Amat);
	gen_b_vec(b_h);

	//set values of helper function input
	(*help_input_ptr).initS(b_h, Amat);
	//ask to start help function	
	cout << "Main calling help function for 1st time" << endl;
	*call_help = 1;
	
	//=====USER CODE before calling help GOES HERE==========
	sleep(.5);


	for(j = 0; j < numReads; j++){
	gettimeofday(&start, NULL);

		//BELOW IS WHERE YOU CALL THE HELPER READ FROM MAIN
		*request_val = 1;
		while(*request_done == 0)
			sleep(.00000001);
		//ABOVE IS WHERE YOU CALL THE HELPER READ FROM MAIN -- now help value(s) is in *help_out

	gettimeofday(&stop, NULL);
	sval = (stop.tv_sec-start.tv_sec)*1000000; //sec to us
	sval = sval + stop.tv_usec-start.tv_usec; //us

	cout << "Time between message request and message receive in us is: " << sval << endl;
	for(i = 0; i < 3; i++)
		cout << "Main update received " << help_out[i] << endl;
	*request_done = 0;
	if(j > 0)  //skip the first call because its bad fro some reason
		sum_times = sum_times+sval;
	sleep(.01);

	}

	
	cout << "Average read time between message request and message received in us is: " << sum_times/(numReads-1) << endl;

	//=======USER code AFTER calling helper goes here======

	cout << "Exiting Main" << endl;
	
	return 1;


}
