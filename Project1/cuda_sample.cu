#include<hip/hip_runtime.h>
#include<iostream>

using namespace std;


__device__ static bool *stop_kernel =0;
__device__ volatile bool *request_read = 0;
__device__ volatile bool *ready_to_read = 0;
__device__ volatile bool *read_complete = 0;

__global__ void dataKernel( int* data){
//this adds a value to a variable stored in global memory
	*data = 3;
	//if(*stop_kernel == 1)
	//	*data = 4;
	}
/*
	while(1){
		if(*stop_kernel == 1){
			*data = 4;
			__syncthreads();
			asm("trap;");
		}
		if(*request_read == 1){
			__syncthreads();
			*ready_to_read = 1;
			while(*read_complete == 0)
			{}
			*request_read = 0;
			*read_complete = 0;
			*data = 5;
		}			


		
	}

}*/


__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr){
	*request_read = 1;

	while(*ready_to_read == 0)
		{}
	*write_2_ptr = *read_in_ptr;
	*read_complete =1;
	*ready_to_read = 0;

}

int main()
{

	int *dVal;
	int size = sizeof(bool);

	//pointer of helper function return	
	int transfered_data;
	int *h_data = &transfered_data;
	int *monitor_data;
	bool k_stop_cmd = 1;
	bool *host_stop_kernel = &k_stop_cmd;

	cout << "k_stop_cmd: " << k_stop_cmd << endl;
	cout << "*host_stop_kernel" << *host_stop_kernel << endl;
	*host_stop_kernel = 1;
	cout << "set *host_stop kernel=1" << endl;
	cout << "k_stop_cmd: " << k_stop_cmd << endl;
	cout << "*host_stop_kernel" << *host_stop_kernel << endl;
	bool *test_value;
	bool *stop_kern_ptr;
	hipError_t cErr;
	//bool *stop_kern_ptr = &stop_kernel;
		
	cErr = hipHostMalloc((void**)&host_stop_kernel, size, hipHostMallocDefault);
	cout << "*host_stop_kernel: " << *host_stop_kernel << endl;	
	cErr = hipMalloc((void**)&stop_kernel, size);

	hipGetSymbolAddress((void**)&stop_kern_ptr, HIP_SYMBOL(stop_kernel));
	cout << "ADDRESs Of stop_kernel = " << stop_kern_ptr << endl;
	//cout << "Dereferenced stop kernel = " << *stop_kern_ptr << endl;
	//cudaMalloc((void**)&dVal, sizeof(int));
	hipMalloc((void**)&dVal, sizeof(int));
		
	//cout << "Cuda Error: " << cErr << endl;	

	cout <<"Trying to Stop Helper Kernel" << endl;
	//cudaMemcpyToSymbol(stop_kernel, host_stop_kernel, sizeof(bool), cudaMemcpyHostToDevice);
	cout << "Copying " << *host_stop_kernel << " from the address: " << host_stop_kernel << "to: " << stop_kern_ptr << endl;
	hipMemcpy(stop_kern_ptr, host_stop_kernel, sizeof(bool), hipMemcpyHostToDevice);
cout <<"COPIED MEM DO DEVICE" << endl;
	cout << "Copying from" << stop_kern_ptr << "to: " << &test_value << endl;
	
	hipMemcpy(&test_value, stop_kern_ptr, sizeof(bool), hipMemcpyDeviceToHost);
	cout << "if stop_kernel in global memory of device then this better be 1: " << *test_value << endl;
	//cudaStreamSynchronize(stream1);
	dataKernel<<<1, 1>>>(dVal);
	hipMemcpy(h_data, dVal, sizeof(int), hipMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;
return 0;


	hipStream_t stream1;
	hipStreamCreate(&stream1);


	hipMalloc(&request_read, sizeof(bool));
	hipMalloc(&read_complete, sizeof(bool));
	hipMalloc(&ready_to_read, sizeof(bool));
	
	hipHostMalloc((void**)&monitor_data, sizeof(int), hipHostMallocDefault);

	cout <<"Launching Monitor Kernel" << endl;
	hipStreamSynchronize(stream1);
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	hipMemcpyAsync(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipStreamSynchronize(stream1);
	
	cout << "Value monitored: "  << *h_data << endl;
	//bool k_stop_cmd = 1;
	//bool *host_stop_kernel = &k_stop_cmd;
	cout <<"Trying to Stop Helper Kernel" << endl;
	hipMemcpyAsync(&stop_kernel, host_stop_kernel, sizeof(bool), hipMemcpyHostToDevice, stream1);
	hipStreamSynchronize(stream1);
	hipMemcpy(h_data, dVal, sizeof(int), hipMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;

	hipFree(dVal);
	hipFree(&stop_kernel);
	hipFree(&request_read);
	hipFree(&read_complete);
	hipFree(&ready_to_read);
	
	return 0;
	/*
cudaMemcpy(&hostArray, dArray, sizeof(int)*numElems, cudaMemcpyDeviceToHost);

	cout << "Values in hostArray: " << endl;
	for(i = 0; i < numElems; i++)
		cout << hostArray[i] << endl;
	cudaFree(dArray);
	return 0;
*/
}
