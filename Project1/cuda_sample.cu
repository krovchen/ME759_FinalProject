#include<hip/hip_runtime.h>
#include<iostream>
#include <unistd.h>

using namespace std;

const int numElems =2;

__global__ void dataKernel( double* data, int nsteps){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x;
	//data[thid] = 0;
	int i = 0;
	bool wait = 1;

	clock_t start = clock64();
	clock_t now;

	while(i < nsteps){
		data[thid] = data[thid]+.1;

		clock_t start = clock64();
		i = i+1;
		while(wait == 1){
			now = clock();
			clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
			if(cycles > 5000)
				wait = 0;
		}		
		wait = 1;
		__syncthreads();
	}	



}


__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr){
	


	*write_2_ptr = *read_in_ptr;

}

int main()
{

			cout <<"Running CUDA init" << endl;

			double *dArray;

			int i = 0;

			//pointer of helper function return	

			double h_data[numElems];
			//double monitor_data[numElems];
			double monitor_data_dev[numElems];

			hipMalloc((void**)&dArray, sizeof(double)*numElems);
			hipMemset(dArray, 0, numElems*sizeof(double));
			hipHostMalloc((void**)&h_data, sizeof(double)*numElems, hipHostMallocDefault);
			hipStream_t stream1;
			hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
			hipMalloc((void**)&monitor_data_dev, sizeof(double)*numElems);


		cout <<"Launching Helper Kernel" << endl;
			//*help_rdy =  help_fcn(*help_input, out);
			dataKernel<<<1,numElems>>>(dArray, 1000);
			sleep(.4);

					cout <<"Launching Monitor Kernel" << endl;
					//cudaStreamSynchronize(stream1);
					monitorKernel<<<1, 1,0, stream1>>>(monitor_data_dev, dArray);
					cout <<"Launching Async Mem Cpy" << endl;
					hipMemcpyAsync(h_data, monitor_data_dev, numElems*sizeof(double), hipMemcpyDeviceToHost, stream1);
					hipStreamSynchronize(stream1);
						for(i = 0; i < numElems; i++)
				cout << "Value copied over: "  << h_data[i] << endl;
					sleep(.3);
							cout <<"Launching Monitor Kernel" << endl;
					//cudaStreamSynchronize(stream1);
					monitorKernel<<<1, 1,0, stream1>>>(monitor_data_dev, dArray);
					cout <<"Launching Async Mem Cpy" << endl;
					hipMemcpyAsync(h_data, monitor_data_dev, numElems*sizeof(double), hipMemcpyDeviceToHost, stream1);
					hipStreamSynchronize(stream1);
						for(i = 0; i < numElems; i++)
				cout << "Value copied over: "  << h_data[i] << endl;



			hipMemcpy(h_data, dArray, sizeof(double)*numElems, hipMemcpyDeviceToHost);
			for(i = 0; i < numElems; i++)
				cout << "Value copied over: "  << h_data[i] << endl;

			hipFree(dArray);
	
			hipFree(monitor_data_dev);
return 0;


	



}
