#include<hip/hip_runtime.h>
#include<iostream>

using namespace std;


__device__ volatile bool *stop_kernel =0;
__device__ volatile bool *request_read = 0;
__device__ volatile bool *ready_to_read = 0;
__device__ volatile bool *read_complete = 0;

__global__ void dataKernel( int* data){
//this adds a value to a variable stored in global memory
	*data = 3;
	if(*stop_kernel == 1)
		*data = 4;
	}
/*
	while(1){
		if(*stop_kernel == 1){
			*data = 4;
			__syncthreads();
			asm("trap;");
		}
		if(*request_read == 1){
			__syncthreads();
			*ready_to_read = 1;
			while(*read_complete == 0)
			{}
			*request_read = 0;
			*read_complete = 0;
			*data = 5;
		}			


		
	}

}*/


__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr){
	*request_read = 1;

	while(*ready_to_read == 0)
		{}
	*write_2_ptr = *read_in_ptr;
	*read_complete =1;
	*ready_to_read = 0;

}

int main()
{

	int *dVal;
	int size = sizeof(bool);

	//pointer of helper function return	
	int transfered_data;
	int *h_data = &transfered_data;
	int *monitor_data;
		bool k_stop_cmd = 1;
	bool *host_stop_kernel = &k_stop_cmd;
	hipError_t cErr;
	//bool *stop_kern_ptr = &stop_kernel;
		
	cErr = hipMalloc((void**)&stop_kernel, size);

	hipMalloc((void**)&dVal, sizeof(int));

		
	//cout << "Cuda Error: " << cErr << endl;	

	cout <<"Trying to Stop Helper Kernel" << endl;
	hipMemcpy(&stop_kernel, host_stop_kernel, sizeof(bool), hipMemcpyHostToDevice);
	//cudaStreamSynchronize(stream1);
	dataKernel<<<1, 1>>>(dVal);
	hipMemcpy(h_data, dVal, sizeof(int), hipMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;
return 0;


	hipStream_t stream1;
	hipStreamCreate(&stream1);


	hipMalloc(&request_read, sizeof(bool));
	hipMalloc(&read_complete, sizeof(bool));
	hipMalloc(&ready_to_read, sizeof(bool));
	
	hipHostMalloc((void**)&monitor_data, sizeof(int), hipHostMallocDefault);

	cout <<"Launching Monitor Kernel" << endl;
	hipStreamSynchronize(stream1);
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	hipMemcpyAsync(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipStreamSynchronize(stream1);
	
	cout << "Value monitored: "  << *h_data << endl;
	//bool k_stop_cmd = 1;
	//bool *host_stop_kernel = &k_stop_cmd;
	cout <<"Trying to Stop Helper Kernel" << endl;
	hipMemcpyAsync(&stop_kernel, host_stop_kernel, sizeof(bool), hipMemcpyHostToDevice, stream1);
	hipStreamSynchronize(stream1);
	hipMemcpy(h_data, dVal, sizeof(int), hipMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;

	hipFree(dVal);
	hipFree(&stop_kernel);
	hipFree(&request_read);
	hipFree(&read_complete);
	hipFree(&ready_to_read);
	
	return 0;
	/*
cudaMemcpy(&hostArray, dArray, sizeof(int)*numElems, cudaMemcpyDeviceToHost);

	cout << "Values in hostArray: " << endl;
	for(i = 0; i < numElems; i++)
		cout << hostArray[i] << endl;
	cudaFree(dArray);
	return 0;
*/
}
