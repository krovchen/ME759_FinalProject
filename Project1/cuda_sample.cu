#include<hip/hip_runtime.h>
#include<iostream>
#include <unistd.h>

using namespace std;


//__device__ static bool *stop_kernel =0;


__global__ void dataKernel( int* data){
//this adds a value to a variable stored in global memory

	*data = 3;
	bool x = 0;
	bool *stop = &x;
	while(1){
		if(*data > 300)
			*data = 0;
		*data = *data+1;
		if(*stop == 1){
			*data = 6;
			__syncthreads();
			asm("trap;");
		}

		
	}

}


__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr){
	


	*write_2_ptr = *read_in_ptr;
	/*if(*rc == 1){
		
		*rc = 0;
	}*/
	/*
	*req_rd = 1;

	while(*r2r == 0)
		{}
	*write_2_ptr = *read_in_ptr;
	*rc =1;
	*r2r = 0;
	*/

}

int main()
{

	int *dVal;
	//int size = sizeof(bool);

	//pointer of helper function return	asdfasdfasdf
	int transfered_data;
	int *h_data = &transfered_data;
	int *monitor_data;
	//bool k_stop_cmd = 1;
	//bool *host_stop_kernel = &k_stop_cmd;

	//bool bool_test = 0;
	//bool *test_value = &bool_test;
	
	//cudaMallocHost((void**)&host_stop_kernel, size);
	//*host_stop_kernel = 1;

	//cudaMalloc((void**)&stop_kernel, size);
	
	//bool *stop_kern_ptr;
	//cudaGetSymbolAddress((void**)&stop_kern_ptr, stop_kernel);

	hipMalloc((void**)&dVal, sizeof(int));
	hipHostMalloc((void**)&monitor_data, sizeof(int), hipHostMallocDefault);	


	hipStream_t stream1;
	hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
	
	dataKernel<<<1, 1>>>(dVal);

	cout <<"Launching Monitor Kernel" << endl;
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	hipMemcpyAsync(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	hipStreamSynchronize(stream1);

	sleep(2);
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	hipMemcpyAsync(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	hipStreamSynchronize(stream1);

sleep(1);

	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	hipMemcpyAsync(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	hipStreamSynchronize(stream1);

sleep(1);


	//cout << "Stopping Kernel " << *host_stop_kernel << endl;
	//cudaMemcpyAsync(stop_kern_ptr, host_stop_kernel, sizeof(bool), cudaMemcpyHostToDevice, stream1);
	//cudaStreamSynchronize(stream1);
	
	//cudaMemcpyAsync(test_value, stop_kern_ptr, sizeof(bool), cudaMemcpyDeviceToHost, stream1);
	
	//cudaStreamSynchronize(stream1);
//cout << "if stop_kernel in global memory of device then this better be 1: " << *test_value << endl;

	hipMemcpy(h_data, dVal, sizeof(int), hipMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;
	hipFree(dVal);
return 0;


	






	
	

	/*
cudaMemcpy(&hostArray, dArray, sizeof(int)*numElems, cudaMemcpyDeviceToHost);

	cout << "Values in hostArray: " << endl;
	for(i = 0; i < numElems; i++)
		cout << hostArray[i] << endl;
	cudaFree(dArray);
	return 0;
*/
}
