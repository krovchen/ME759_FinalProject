#include<hip/hip_runtime.h>
#include<iostream>

using namespace std;


__device__ static bool *stop_kernel =0;
__device__ static bool *request_read = 0;
__device__ static bool *ready_to_read = 0;
__device__ static bool *read_complete = 0;

__global__ void dataKernel( int* data, bool* stop, bool* req_red, bool *r2r, bool *rdc){
//this adds a value to a variable stored in global memory

	*data = 3;
	if(*stop == 1)
		*data = 4;
	

	while(1){
		if(*stop == 1){
			*data = 6;
			__syncthreads();
			asm("trap;");
		}
		if(*req_red == 1){
			__syncthreads();
			*ready_to_read = 1;
			while(*rdc == 0)
			{}
			*r2r = 0;
			*rdc = 0;
			*data = 5;
		}			


		
	}

}


__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr, bool* req_rd, bool *r2r, bool *rc){
	*req_rd = 1;

	while(*r2r == 0)
		{}
	*write_2_ptr = *read_in_ptr;
	*rc =1;
	*r2r = 0;

}

int main()
{

	int *dVal;
	int size = sizeof(bool);

	//pointer of helper function return	
	int transfered_data;
	int *h_data = &transfered_data;
	int *monitor_data;
	bool k_stop_cmd = 1;
	bool *host_stop_kernel = &k_stop_cmd;

	bool bool_test = 0;
	bool *test_value = &bool_test;
	bool *stop_kern_ptr;
	bool *request_read_ptr;
	bool *read_to_read_ptr;
	bool *read_complete_ptr;
	
	hipHostMalloc((void**)&host_stop_kernel, size, hipHostMallocDefault);
	*host_stop_kernel = 1;

	hipMalloc((void**)&stop_kernel, size);
	hipMalloc((void**)&request_read, size);
	hipMalloc((void**)&ready_to_read, size);
	hipMalloc((void**)&read_complete, size);

	hipGetSymbolAddress((void**)&stop_kern_ptr, HIP_SYMBOL(stop_kernel));
	hipGetSymbolAddress((void**)&request_read_ptr, HIP_SYMBOL(request_read));
	hipGetSymbolAddress((void**)&read_to_read_ptr, HIP_SYMBOL(ready_to_read));
	hipGetSymbolAddress((void**)&read_complete_ptr, HIP_SYMBOL(read_complete));

	cout << "ADDRESs Of stop_kernel = " << stop_kern_ptr << endl;
	//cout << "Dereferenced stop kernel = " << *stop_kern_ptr << endl;
	//cudaMalloc((void**)&dVal, sizeof(int));
	hipMalloc((void**)&dVal, sizeof(int));
		
	//cout << "Cuda Error: " << cErr << endl;	

	
	//cudaMemcpyToSymbol(stop_kernel, host_stop_kernel, sizeof(bool), cudaMemcpyHostToDevice);
	/*cout << "Copying " << *host_stop_kernel << " from the address: " << host_stop_kernel << "to: " << stop_kern_ptr << endl;
	cudaMemcpy(stop_kern_ptr, host_stop_kernel, sizeof(bool), cudaMemcpyHostToDevice);
cout <<"COPIED MEM DO DEVICE" << endl;
	cout << "Copying from" << stop_kern_ptr << "to: " << &test_value << endl;
	
	cudaMemcpy(test_value, stop_kern_ptr, sizeof(bool), cudaMemcpyDeviceToHost);
	cout << "Test value = : " << test_value << endl;
	cout << "if stop_kernel in global memory of device then this better be 1: " << *test_value << endl;*/
	//cudaStreamSynchronize(stream1);
	dataKernel<<<1, 1>>>(dVal, stop_kern_ptr, request_read_ptr, read_to_read_ptr, read_complete_ptr);



	hipStream_t stream1;
	hipStreamCreate(&stream1);
	hipHostMalloc((void**)&monitor_data, sizeof(int), hipHostMallocDefault);
	cout <<"Launching Monitor Kernel" << endl;
	hipStreamSynchronize(stream1);
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal, request_read_ptr, read_to_read_ptr, read_complete_ptr);
	cout <<"Launching Async Mem Cpy" << endl;
	hipMemcpyAsync(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipStreamSynchronize(stream1);
	cout << "Value monitored: "  << *h_data << endl;
cout << "Copying " << *host_stop_kernel << " from the address: " << host_stop_kernel << "to: " << stop_kern_ptr << endl;
	hipMemcpy(stop_kern_ptr, host_stop_kernel, sizeof(bool), hipMemcpyHostToDevice);
cout <<"COPIED MEM DO DEVICE" << endl;
	hipMemcpy(h_data, dVal, sizeof(int), hipMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;
return 0;





	hipMalloc(&request_read, sizeof(bool));
	hipMalloc(&read_complete, sizeof(bool));
	hipMalloc(&ready_to_read, sizeof(bool));
	
	


	cout <<"Launching Async Mem Cpy" << endl;
	hipMemcpyAsync(h_data, monitor_data, sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipStreamSynchronize(stream1);
	
	cout << "Value monitored: "  << *h_data << endl;
	//bool k_stop_cmd = 1;
	//bool *host_stop_kernel = &k_stop_cmd;
	cout <<"Trying to Stop Helper Kernel" << endl;
	hipMemcpyAsync(&stop_kernel, host_stop_kernel, sizeof(bool), hipMemcpyHostToDevice, stream1);
	hipStreamSynchronize(stream1);
	hipMemcpy(h_data, dVal, sizeof(int), hipMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;

	hipFree(dVal);
	hipFree(&stop_kernel);
	hipFree(&request_read);
	hipFree(&read_complete);
	hipFree(&ready_to_read);
	
	return 0;
	/*
cudaMemcpy(&hostArray, dArray, sizeof(int)*numElems, cudaMemcpyDeviceToHost);

	cout << "Values in hostArray: " << endl;
	for(i = 0; i < numElems; i++)
		cout << hostArray[i] << endl;
	cudaFree(dArray);
	return 0;
*/
}
