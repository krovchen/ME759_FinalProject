#include<hip/hip_runtime.h>
#include<iostream>
#include <unistd.h>

using namespace std;

const int numElems =2;

__global__ void dataKernel( double* data, int nsteps){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x;
	//data[thid] = 0;
	int i = 0;
	bool wait = 1;

	clock_t start = clock64();
	clock_t now;

	while(i < nsteps){
		data[thid] = data[thid]+.1;

		clock_t start = clock64();
		i = i+1;
		while(wait == 1){
			now = clock();
			clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
			if(cycles > 5000)
				wait = 0;
		}		
		wait = 1;
		__syncthreads();
	}	



}


__global__ void monitorKernel(int * write_2_ptr,  int * read_in_ptr){
	


	*write_2_ptr = *read_in_ptr;

}

int main()
{

			cout <<"Running CUDA init" << endl;

			double *dArray;

			int i = 0;

			//pointer of helper function return	

			double h_data[numElems];


			hipMalloc((void**)&dArray, sizeof(double)*numElems);
			hipMemset(dArray, 0, numElems*sizeof(double));
			hipHostMalloc((void**)&h_data, sizeof(double)*numElems, hipHostMallocDefault);



		cout <<"Launching Helper Kernel" << endl;
			//*help_rdy =  help_fcn(*help_input, out);
			dataKernel<<<1,numElems>>>(dArray, 1000);


			hipMemcpy(h_data, dArray, sizeof(double)*numElems, hipMemcpyDeviceToHost);
			for(i = 0; i < numElems; i++)
				cout << "Value copied over: "  << h_data[i] << endl;

			hipFree(dArray);
	

return 0;


	



}
