#include<hip/hip_runtime.h>
#include<iostream>
#include <unistd.h>
#include <math.h>
#include <stdlib.h>

using namespace std;


__global__ void dataKernel(double* data, double* A, double* B, int nsteps);
__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr);


__global__ void Muldev(double* A, double* B, double* C, int nRows)
{

	extern __shared__ double ptr[];


	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int aBegin = nRows*blockDim.x*by;
	int aEnd = aBegin + nRows-1;

	int aStep = blockDim.x;
	
	int bBegin = blockDim.x*bx;
	int bStep = blockDim.x*nRows;

	double Csub = 0;
	
	double* As = &ptr[0];
	double* Bs = &ptr[blockDim.x*blockDim.x];

	

	int a;
	int b;
	int k;

	for(a = aBegin, b = bBegin; a<=aEnd; a+=aStep, b+=bStep){
		
		As[ty*blockDim.x+tx] = A[a+nRows*ty+tx];

		Bs[ty*blockDim.x+tx] = B[b+nRows*ty+tx];
		

		__syncthreads();
		__threadfence_block();
		
		for(k = 0; k < blockDim.x; ++k){
			Csub+=As[ty*blockDim.x+k]*Bs[k*blockDim.x+tx];}
			//Csub+=As[tx*blockDim.x+k]*Bs[k*blockDim.x+ty];}
			
		__syncthreads();
	}

	int c = nRows*blockDim.x*by+blockDim.x*bx;
	C[c+nRows*ty+tx] = Csub;
		


}


int main()
{


	double hA[4] = {1/150, .1/150, -.1/150, -1/150};
	double hB[4] = {.6, -.1, .6, 1.95};
	double hC[4];
	double* dA;
	double* dB;
	double* dC;
	int nRows = 2;
	int TileSize = 1;

	int size = 4*sizeof(double);


	hipMalloc((void**)&dA, size);
	hipMalloc((void**)&dB, size);
	hipMalloc((void**)&dC, size);
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

	dim3 dimBlock(TileSize, TileSize);
	dim3 dimGrid(nRows/TileSize, nRows/TileSize);

	
	Muldev<<<dimGrid, dimBlock, sizeof(double)*TileSize*TileSize*TileSize*TileSize>>>(dA, dB, dC, nRows);
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	int i = 0;
	for(i = 0; i < 4; i++)
		cout << "hC: " << hC[i] << endl;


	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

return 0;
}
	/*int *dVal;
	int size = sizeof(bool);

	//pointer of helper function return	asdfasdfasdf
	int transfered_data;
	int *h_data = &transfered_data;
	int *monitor_data;
	bool k_stop_cmd = 1;
	bool *host_stop_kernel = &k_stop_cmd;

	bool bool_test = 0;
	bool *test_value = &bool_test;
	
	cudaMallocHost((void**)&host_stop_kernel, size);
	*host_stop_kernel = 1;

	cudaMalloc((void**)&stop_kernel, size);
	
	bool *stop_kern_ptr;
	cudaGetSymbolAddress((void**)&stop_kern_ptr, stop_kernel);

	cudaMalloc((void**)&dVal, sizeof(int));
	cudaMallocHost((void**)&monitor_data, sizeof(int));	


	cudaStream_t stream1;
	cudaStreamCreateWithFlags(&stream1, cudaStreamNonBlocking);
	
	dataKernel<<<1, 1>>>(dVal, stop_kern_ptr);

	cout <<"Launching Monitor Kernel" << endl;
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	cudaMemcpyAsync(h_data, monitor_data, sizeof(int), cudaMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	cudaStreamSynchronize(stream1);

	sleep(2);
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	cudaMemcpyAsync(h_data, monitor_data, sizeof(int), cudaMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	cudaStreamSynchronize(stream1);

sleep(1);

	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	cudaMemcpyAsync(h_data, monitor_data, sizeof(int), cudaMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	cudaStreamSynchronize(stream1);

sleep(1);


	cout << "Stopping Kernel " << *host_stop_kernel << endl;
	cudaMemcpyAsync(stop_kern_ptr, host_stop_kernel, sizeof(bool), cudaMemcpyHostToDevice, stream1);
	cudaStreamSynchronize(stream1);
	
	cudaMemcpyAsync(test_value, stop_kern_ptr, sizeof(bool), cudaMemcpyDeviceToHost, stream1);
	
	cudaStreamSynchronize(stream1);
cout << "if stop_kernel in global memory of device then this better be 1: " << *test_value << endl;

	cudaMemcpy(h_data, dVal, sizeof(int), cudaMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;
return 0;*/





__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr){

	*write_2_ptr = *read_in_ptr;
}


__global__ void dataKernel(double* data, double* A, double* B, int nsteps){
//this adds a value to a variable stored in global memory

	int i = 0;

	int thid = threadIdx.x;

	while(i < nsteps){
		//u1[thid] = u1[thid]*u1[thid]+5;

		
	}

}


