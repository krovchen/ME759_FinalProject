#include<hip/hip_runtime.h>
#include<iostream>
#include <unistd.h>
#include <math.h>
#include <stdlib.h>

using namespace std;


__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr);


__device__ void MatrixAddKernel(double* Melems, double* Nelems, double* Pelems)
{
 
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	int i = 0;
	int shift = thid;
	for(i = 0; i < 1; ++i)
	Pelems[shift+i] = Melems[shift+i]+Nelems[shift+i];

}


__device__ void Muldev(double* A, double* B, double* C, int nRows)
{

	extern __shared__ double ptr[];


	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int aBegin = nRows*blockDim.x*by;
	int aEnd = aBegin + nRows-1;

	int aStep = blockDim.x;
	
	int bBegin = blockDim.x*bx;
	int bStep = blockDim.x*nRows;

	double Csub = 0;
	
	double* As = &ptr[0];
	double* Bs = &ptr[blockDim.x*blockDim.x];

	

	int a;
	int b;
	int k;

	for(a = aBegin, b = bBegin; a<=aEnd; a+=aStep, b+=bStep){
		
		As[ty*blockDim.x+tx] = A[a+nRows*ty+tx];

		Bs[ty*blockDim.x+tx] = B[b+nRows*ty+tx];
		

		__syncthreads();
		__threadfence_block();
		
		for(k = 0; k < blockDim.x; ++k){
			Csub+=As[ty*blockDim.x+k]*Bs[k*blockDim.x+tx];}
			//Csub+=As[tx*blockDim.x+k]*Bs[k*blockDim.x+ty];}
			
		__syncthreads();
	}

	int c = nRows*blockDim.x*by+blockDim.x*bx;
	C[c+nRows*ty+tx] = Csub;
		


}


__global__ void dataKernel(double* data, double* A, double* B, int nsteps, double *temp1, double *temp2, double* temp3){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	temp3[thid] = sin(data[thid]);
	__syncthreads();
	if(thid == 0){
		Muldev(data, data, temp1, 2);
		Muldev(B, data, temp2, 2);
		Muldev(A, temp3, temp3, 2);
		MatrixAddKernel(temp1, temp2, data);
		MatrixAddKernel(data, temp3, data);

	}
}


int main()
{


	double hA[4] = {1, 2, 3, 4};
	double hB[4] = {1, 2, 3, 4};
	double hC[4] = {.3, .3, -.5, -.25};
	double* dA;
	double* dB;
	double* dC;
	double *temp1, *temp2, *temp3;
	int nRows = 2;
	int TileSize = 1;

	int size = 4*sizeof(double);


	hipMalloc((void**)&dA, size);
	hipMalloc((void**)&dB, size);
	hipMalloc((void**)&dC, size);
	hipMalloc((void**)&temp1, size);
	hipMalloc((void**)&temp2, size);
	hipMalloc((void**)&temp3, size);
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
	hipMemcpy(dC, hC, size, hipMemcpyHostToDevice);

	dim3 dimBlock(TileSize, TileSize);
	dim3 dimGrid(nRows/TileSize, nRows/TileSize);

	
	dataKernel<<<dimGrid, dimBlock, sizeof(double)*TileSize*TileSize*TileSize*TileSize>>>(dC, dA, dB, 1000, temp1, temp2, temp3);
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	int i = 0;
	for(i = 0; i < 4; i++)
		cout << "hC: " << hC[i] << endl;


	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

return 0;
}
	/*int *dVal;
	int size = sizeof(bool);

	//pointer of helper function return	asdfasdfasdf
	int transfered_data;
	int *h_data = &transfered_data;
	int *monitor_data;
	bool k_stop_cmd = 1;
	bool *host_stop_kernel = &k_stop_cmd;

	bool bool_test = 0;
	bool *test_value = &bool_test;
	
	cudaMallocHost((void**)&host_stop_kernel, size);
	*host_stop_kernel = 1;

	cudaMalloc((void**)&stop_kernel, size);
	
	bool *stop_kern_ptr;
	cudaGetSymbolAddress((void**)&stop_kern_ptr, stop_kernel);

	cudaMalloc((void**)&dVal, sizeof(int));
	cudaMallocHost((void**)&monitor_data, sizeof(int));	


	cudaStream_t stream1;
	cudaStreamCreateWithFlags(&stream1, cudaStreamNonBlocking);
	
	dataKernel<<<1, 1>>>(dVal, stop_kern_ptr);

	cout <<"Launching Monitor Kernel" << endl;
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	cudaMemcpyAsync(h_data, monitor_data, sizeof(int), cudaMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	cudaStreamSynchronize(stream1);

	sleep(2);
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	cudaMemcpyAsync(h_data, monitor_data, sizeof(int), cudaMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	cudaStreamSynchronize(stream1);

sleep(1);

	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	cudaMemcpyAsync(h_data, monitor_data, sizeof(int), cudaMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	cudaStreamSynchronize(stream1);

sleep(1);


	cout << "Stopping Kernel " << *host_stop_kernel << endl;
	cudaMemcpyAsync(stop_kern_ptr, host_stop_kernel, sizeof(bool), cudaMemcpyHostToDevice, stream1);
	cudaStreamSynchronize(stream1);
	
	cudaMemcpyAsync(test_value, stop_kern_ptr, sizeof(bool), cudaMemcpyDeviceToHost, stream1);
	
	cudaStreamSynchronize(stream1);
cout << "if stop_kernel in global memory of device then this better be 1: " << *test_value << endl;

	cudaMemcpy(h_data, dVal, sizeof(int), cudaMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;
return 0;*/





__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr){

	*write_2_ptr = *read_in_ptr;
}


