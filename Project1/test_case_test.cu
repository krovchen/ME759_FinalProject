#include<hip/hip_runtime.h>
#include<iostream>
#include <unistd.h>
#include <math.h>
#include <stdlib.h>

using namespace std;


__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr);


__device__ void MatrixAddKernel(double* Melems, double* Nelems, double* Pelems)
{
 
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	int i = 0;
	int shift = thid;
	for(i = 0; i < 1; ++i)
	Pelems[shift+i] = Melems[shift+i]+Nelems[shift+i];

}


__device__ void Muldev(double* A, double* B, double* C, int nRows)
{

	extern __shared__ double ptr[];


	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int aBegin = nRows*blockDim.x*by;
	int aEnd = aBegin + nRows-1;

	int aStep = blockDim.x;
	
	int bBegin = blockDim.x*bx;
	int bStep = blockDim.x*nRows;

	double Csub = 0;
	
	double* As = &ptr[0];
	double* Bs = &ptr[blockDim.x*blockDim.x];

	

	int a;
	int b;
	int k;

	for(a = aBegin, b = bBegin; a<=aEnd; a+=aStep, b+=bStep){
		
		As[ty*blockDim.x+tx] = A[a+nRows*ty+tx];

		Bs[ty*blockDim.x+tx] = B[b+nRows*ty+tx];
		

		__syncthreads();
		__threadfence_block();
		
		for(k = 0; k < blockDim.x; ++k){
			Csub+=As[ty*blockDim.x+k]*Bs[k*blockDim.x+tx];}
			//Csub+=As[tx*blockDim.x+k]*Bs[k*blockDim.x+ty];}
			
		__syncthreads();
	}

	int c = nRows*blockDim.x*by+blockDim.x*bx;
	C[c+nRows*ty+tx] = Csub;
		


}


__global__ void dataKernel(double* data, double* A, double* B, int nsteps, double *temp1, double *temp2, double* temp3){
//this adds a value to a variable stored in global memory


	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int thid = tx + blockDim.x*ty;

	int i = 0;
	while(i < nsteps)
	{
		temp3[thid] = sin(data[thid]);
		__syncthreads();
	
		Muldev(data, data, temp1, 2);
		__syncthreads();
		Muldev(B, data, temp2, 2);
		__syncthreads();
		Muldev(A, temp3, temp3, 2);
		__syncthreads();


		data[thid] = temp1[thid]+temp2[thid]+temp3[thid];
		__syncthreads();
		i = i+1;
	}
}


int main(int argc, char** argv)
{


	double hA[4] = {.6, -.1, .6, 1.95};
	double hB[4] = {1/150, .1/150, -.1/150, -1/150};
	double hC[4] = {.3, .3, -.5, -.25};
	double* dA;
	double* dB;
	double* dC;
	double *temp1, *temp2, *temp3;
	int nRows = 2;
	int TileSize = 2;

	int size = 4*sizeof(double);
	int nSteps = atoi(argv[1]);

	hipMalloc((void**)&dA, size);
	hipMalloc((void**)&dB, size);
	hipMalloc((void**)&dC, size);
	hipMalloc((void**)&temp1, size);
	hipMalloc((void**)&temp2, size);
	hipMalloc((void**)&temp3, size);
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
	hipMemcpy(dC, hC, size, hipMemcpyHostToDevice);

	dim3 dimBlock(TileSize, TileSize);
	dim3 dimGrid(nRows/TileSize, nRows/TileSize);

	double *monitor_data;
	double *h_data;
	hipMalloc((void**)&monitor_data, sizeof(double));
	hipHostMalloc((void**)&h_data, sizeof(double), hipHostMallocDefault);
	hipStream_t stream1;
	hipStreamCreate(&stream1);	


	dataKernel<<<dimGrid, dimBlock, sizeof(double)*TileSize*TileSize*TileSize*TileSize>>>(dC, dA, dB, nSteps, temp1, temp2, temp3);
	sleep(.0001);
		cout <<"Launching Monitor Kernel" << endl;
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, &dC[1]);
	cout <<"Launching Async Mem Cpy" << endl;
	hipMemcpyAsync(h_data, monitor_data, sizeof(double), hipMemcpyDeviceToHost, stream1);

	hipStreamSynchronize(stream1);
	cout << "Value monitored over: "  << *h_data*100 << endl;
	sleep(.0001);
		cout <<"Launching Monitor Kernel" << endl;
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, &dC[1]);
	cout <<"Launching Async Mem Cpy" << endl;
	hipMemcpyAsync(h_data, monitor_data, sizeof(double), hipMemcpyDeviceToHost, stream1);

	hipStreamSynchronize(stream1);
		cout << "Value monitored over: "  << *h_data*100 << endl;
	sleep(1);
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	int i = 0;
	for(i = 0; i < 4; i++)
		cout << "hC: " << hC[i]*100 << endl;


	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

return 0;
}
	/*int *dVal;
	int size = sizeof(bool);

	//pointer of helper function return	asdfasdfasdf
	int transfered_data;
	int *h_data = &transfered_data;
	int *monitor_data;
	bool k_stop_cmd = 1;
	bool *host_stop_kernel = &k_stop_cmd;

	bool bool_test = 0;
	bool *test_value = &bool_test;
	
	cudaMallocHost((void**)&host_stop_kernel, size);
	*host_stop_kernel = 1;

	cudaMalloc((void**)&stop_kernel, size);
	
	bool *stop_kern_ptr;
	cudaGetSymbolAddress((void**)&stop_kern_ptr, stop_kernel);

	cudaMalloc((void**)&dVal, sizeof(int));
	cudaMallocHost((void**)&monitor_data, sizeof(int));	


	cudaStream_t stream1;
	cudaStreamCreateWithFlags(&stream1, cudaStreamNonBlocking);
	
	dataKernel<<<1, 1>>>(dVal, stop_kern_ptr);

	cout <<"Launching Monitor Kernel" << endl;
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	cudaMemcpyAsync(h_data, monitor_data, sizeof(int), cudaMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	cudaStreamSynchronize(stream1);

	sleep(2);
	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	cudaMemcpyAsync(h_data, monitor_data, sizeof(int), cudaMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	cudaStreamSynchronize(stream1);

sleep(1);

	monitorKernel<<<1, 1,0, stream1>>>(monitor_data, dVal);
	cout <<"Launching Async Mem Cpy" << endl;
	cudaMemcpyAsync(h_data, monitor_data, sizeof(int), cudaMemcpyDeviceToHost, stream1);
	cout << "Value monitored over: "  << *h_data << endl;
	cudaStreamSynchronize(stream1);

sleep(1);


	cout << "Stopping Kernel " << *host_stop_kernel << endl;
	cudaMemcpyAsync(stop_kern_ptr, host_stop_kernel, sizeof(bool), cudaMemcpyHostToDevice, stream1);
	cudaStreamSynchronize(stream1);
	
	cudaMemcpyAsync(test_value, stop_kern_ptr, sizeof(bool), cudaMemcpyDeviceToHost, stream1);
	
	cudaStreamSynchronize(stream1);
cout << "if stop_kernel in global memory of device then this better be 1: " << *test_value << endl;

	cudaMemcpy(h_data, dVal, sizeof(int), cudaMemcpyDeviceToHost);
	cout << "Value copied over: "  << *h_data << endl;
return 0;*/





__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr){

	*write_2_ptr = *read_in_ptr;
}


