#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include "omp.h"
#include <unistd.h>
#include <sys/time.h>
#include "test_kernels.h"
#include "main_fcn.h"


using namespace std;


__global__ void monitorKernel(double * write_2_ptr,  double * read_in_ptr){

	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	write_2_ptr[thid] = read_in_ptr[thid];


}





__global__ void dataKernel( double* data, double nsteps){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	//data[thid] = 0;
	double i = 0;
	//bool wait = 1;

	//clock_t start = clock64();
	//clock_t now;

	while(i < nsteps){
		data[thid] = data[thid]+.00001*thid;
		i=i+1;
	
	}
		/*start = clock64();
		i = i+1;
		while(wait == 1){
			now = clock();
			clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
			if(cycles > 5000)
				wait = 0;
		}		
		wait = 1;
		
	}	*/



}


bool help_fcn(help_input_from_main help_input, double* out, volatile bool* kernl_rdy){
	//int j = 1;
	int i = 0;
	int j = 0;
	for(j = 0; j < iterations; j++){
		for(i = 0; i < numElems; i++){
			if(i == 0)
				out[i] = i;
			out[i] = out[i]+100/iterations;
	
		}
	//sleep(1);
	}
	return 1;
}



bool main_fcn(ctrl_flags CF, double* help_out, help_input_from_main* help_input_ptr)
{	
	struct timeval stop, start;
	bool *call_help = CF.call_help;
	//volatile bool *help_rdy = CF.help_rdy;
	volatile bool *request_val = CF.request_val;
	volatile bool *request_done = CF.request_done;

	//initialize data for input to helper function
	double inp1[N] = {4};
	int i = 0;
	int numReads = 10;
	double sval;
	double sum_times = 0;
	int j = 0;

	//set values of helper function input
	(*help_input_ptr).initS(inp1);
	//ask to start help function	
	cout << "Main calling help function for 1st time" << endl;
	*call_help = 1;
	
	//=====USER CODE before calling help GOES HERE==========
	sleep(.1);


	for(j = 0; j < numReads; j++){
	gettimeofday(&start, NULL);

		//BELOW IS WHERE YOU CALL THE HELPER READ FROM MAIN
		*request_val = 1;
		while(*request_done == 0)
			sleep(.00000001);
		//ABOVE IS WHERE YOU CALL THE HELPER READ FROM MAIN -- now help value(s) is in *help_out

	gettimeofday(&stop, NULL);
	sval = (stop.tv_sec-start.tv_sec)*1000000; //sec to us
	sval = sval + stop.tv_usec-start.tv_usec; //us

	cout << "Time between message request and message receive in us is: " << sval << endl;
	for(i = 0; i < 3; i++)
		cout << "Main update received " << help_out[i] << endl;
	*request_done = 0;
	if(j > 0)  //skip the first call because its bad fro some reason
		sum_times = sum_times+sval;
	sleep(.2);

	}

	
	cout << "Average read time between message request and message received in us is: " << sum_times/(numReads-1) << endl;

	//=======USER code AFTER calling helper goes here======

	cout << "Exiting Main" << endl;
	
	return 1;


}
