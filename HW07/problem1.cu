#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


using namespace std;

int* read_array(const char* filename, int len) {
	int *x = (int*) malloc(len * sizeof(int));
	FILE *fp = fopen(filename, "r");
	for (int i = 0; i < len; i++) {
		fscanf(fp, "%d", &x[i]);
	}
	fclose(fp);
	return x;
}

void matxvec(int *hA, int *hB, int *hC, int rowWidth, int colWidth){
	int i = 0;
	int j = 0;

	for(i=0; i < colWidth; i++){
		int locsum = 0;
		for(j = 0; j < rowWidth; j++){
			locsum = locsum + hA[i*rowWidth+j]*hB[j];
		}		
		hC[i] = locsum;
	}


}


__global__ void multKernel(int* dA, int* dB, int* dC, int rowWidth, const int colWidth){
	//__shared__ int B[rowWidth];
	//B = dB;
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	int locsum = 0;

	__shared__ int dCshare[16];
	if(threadIdx.x == 0)
		dCshare[blockIdx.x] = 0;
	__syncthreads();
	__threadfence_block();
	if(thid <= rowWidth*colWidth){
		//printf("%s %d\n", "dA: ", dA[thid]);
		
		locsum = dA[thid]*dB[threadIdx.x];

		atomicAdd(&dCshare[blockIdx.x], locsum);

		
	}

	
	if(threadIdx.x == 0)
		dC[blockIdx.x] = dCshare[blockIdx.x];
	
	
}


int main(int argc, char *argv[]) {
	if (argc != 1) {
		printf("Invalid argument Usage: ./problem1");
		return -1;
	}

	const int rowWidth=32;
        const int colWidth=16;
	//int i = 0;	
	int *hA = read_array("inputA.inp",rowWidth*colWidth );
	int *hB = read_array("inputB.inp", rowWidth);
	int *hC = (int*) malloc(colWidth * sizeof(int));
	int *refC = (int*) malloc(colWidth * sizeof(int));
	// TODO - allocate host memory for refC (you have to figure out how much)
	// The skeleton currently segfaults because refC is accessed without allocation++

	// TODO do a reference host implementation (Ch) here. ie populate answer in refC
	matxvec(hA, hB, refC, rowWidth, colWidth);


	int *dA, *dB, *dC;
	// TODO allocate device memory for dA,dB and dC
	// TODO copy data from host to GPU 
	hipMalloc((void**)&dA, sizeof(int)*rowWidth*colWidth);
	hipMemcpy(dA, hA, sizeof(int)*rowWidth*colWidth, hipMemcpyHostToDevice);

	hipMalloc((void**)&dB, sizeof(int)*rowWidth);
	hipMemcpy(dB, hB, sizeof(int)*rowWidth, hipMemcpyHostToDevice);

	hipMalloc((void**)&dC, sizeof(int)*colWidth);
	hipMemset(dC, 0, sizeof(int)*colWidth);


	// TODO call your kernel
	multKernel<<<16, 32, 2*sizeof(int)>>>(dA, dB, dC, rowWidth, colWidth);

	// TODO copyback results
	hipMemcpy(hC, dC, sizeof(int)*colWidth, hipMemcpyDeviceToHost);



	float Error=0;

	for(int i=0;i<colWidth;i++)
		Error+=(hC[i]-refC[i])*(hC[i]-refC[i]);
	printf("%f\n%d\n",sqrt(Error),hC[colWidth-1]);

	free(refC);
	free(hB);
	free(hA);
	free(hC);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);


	return 0;
}
