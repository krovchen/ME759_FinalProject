#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


using namespace std;

int* read_array(const char* filename, int len) {
	int *x = (int*) malloc(len * sizeof(int));
	FILE *fp = fopen(filename, "r");
	for (int i = 0; i < len; i++) {
		fscanf(fp, "%d", &x[i]);
	}
	fclose(fp);
	return x;
}

void matxvec(int *hA, int *hB, int *hC, int rowWidth, int colWidth){
	int i = 0;
	int j = 0;

	for(i=0; i < colWidth; i++){
		int locsum = 0;
		for(j = 0; j < rowWidth; j++){
			locsum = locsum + hA[i*rowWidth+j]*hB[j];
		}		
		hC[i] = locsum;
	}


}


__global__ void multKernel(int* dA, int* dB, int* dC, int rowWidth, int colWidth){
	//__shared__ int B[rowWidth];
	//B = dB;
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	int locsum = 0;
	if(thid < rowWidth*colWidth){
		locsum = dA[thid]*dB[threadIdx.x];
		dC[thid] += locsum;
	}
}


int main(int argc, char *argv[]) {
	if (argc != 1) {
		printf("Invalid argument Usage: ./problem1");
		return -1;
	}

	const int rowWidth=4;//32;
        const int colWidth=2;//16;
	int i = 0;	
	int *hA = read_array("inputA1.inp",rowWidth*colWidth );
	int *hB = read_array("inputB1.inp", rowWidth);
	int *hC = (int*) malloc(colWidth * sizeof(int));
	int *refC = (int*) malloc(colWidth * sizeof(int));
	// TODO - allocate host memory for refC (you have to figure out how much)
	// The skeleton currently segfaults because refC is accessed without allocation

	// TODO do a reference host implementation (Ch) here. ie populate answer in refC
	matxvec(hA, hB, refC, rowWidth, colWidth);



	int *dA, *dB, *dC;
	// TODO allocate device memory for dA,dB and dC
	// TODO copy data from host to GPU 
	hipMalloc((void**)&dA, sizeof(int)*rowWidth*colWidth);
	hipMemcpy(dA, hA, sizeof(int)*rowWidth*colWidth, hipMemcpyHostToDevice);

	hipMalloc((void**)&dB, sizeof(int)*rowWidth);
	hipMemcpy(dB, hC, sizeof(int)*rowWidth, hipMemcpyHostToDevice);

	hipMalloc((void**)&dC, sizeof(int)*colWidth);
	hipMemset(dC, 0, sizeof(int)*colWidth);


	// TODO call your kernel
	multKernel<<<colWidth, rowWidth>>>(dA, dB, dC, rowWidth, colWidth);

	// TODO copyback results
	hipMemcpy(hC, dC, sizeof(int)*colWidth, hipMemcpyDeviceToHost);

	float Error=0;

	for(int i=0;i<colWidth;i++)
		Error+=(hC[i]-refC[i])*(hC[i]-refC[i]);
	printf("%f\n%d",sqrt(Error),hC[colWidth-1]);

	free(refC);
	free(hB);
	free(hA);
	free(hC);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);


	return 0;
}
