#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// includes, project

// includes, kernels

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

double* read_array(const char* filename, int len) {
	double *x = (double*) malloc(len * sizeof(double));
	FILE *fp = fopen(filename, "r");
	for (int i = 0; i < len; i++) {
		fscanf(fp, "%lf", &x[i]);
	}
	fclose(fp);
	return x;
}

void computeOnDevice(double* hA,double* hB, double* hC, int nRows, int tileSize, float* incTime );

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, char** argv) 
{
	if(argc!=2)
	{
		printf("Usage: ./problem2 N\n");
		return 0;
	}
	int nRows = 1024;
	int num_elements = nRows*nRows;
	int tileSize = atoi(argv[1]);  //change this for scaling analysis
	float incTime=0; // Time for GPU
	double* hA = read_array("inputA.inp",num_elements);
	double* hB = read_array("inputB.inp",num_elements);
	double* hC = (double*) malloc(num_elements * sizeof(double));

	// **===-------- Modify the body of this function -----------===**
	computeOnDevice( hA, hB,hC, nRows, tileSize, &incTime);
	// **===-----------------------------------------------------------===**


	printf("%f\n%f\n%d\n",hC[num_elements-1],incTime,tileSize);
	// cleanup memory
	free(hA);
	free(hB);
	free(hC);

	return 0;
}



__global__ void Muldev(double* A, double* B, double* C, int nRows)
{

	extern __shared__ double ptr[];


	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int aBegin = nRows*blockDim.x*by;
	int aEnd = aBegin + nRows-1;

	int aStep = blockDim.x;
	
	int bBegin = blockDim.x*bx;
	int bStep = blockDim.x*nRows;

	double Csub = 0;
	
	double* As = &ptr[0];
	double* Bs = &ptr[blockDim.x*blockDim.x];

	

	int a;
	int b;
	int k;

	for(a = aBegin, b = bBegin; a<=aEnd; a+=aStep, b+=bStep){
		
		As[ty*blockDim.x+tx] = A[a+nRows*ty+tx];

		Bs[ty*blockDim.x+tx] = B[b+nRows*ty+tx];
		

		__syncthreads();
		__threadfence_block();
		
		for(k = 0; k < blockDim.x; ++k)
			

			Csub+=As[ty*blockDim.x+k]*Bs[k*blockDim.x+tx];
			
		__syncthreads();
	}

	int c = nRows*blockDim.x*by+blockDim.x*bx;
	C[c+nRows*ty+tx] = Csub;
		


}

void computeOnDevice(double* hA,double* hB, double* hC, int nRows, int TileSize, float* incTime)
{
	double* dA;
	double* dB;
	double* dC;
	hipEvent_t startEvent_inc;
	hipEvent_t stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);

	int size = nRows*nRows*sizeof(double);

	hipEventRecord(startEvent_inc,0);

	hipMalloc((void**)&dA, size);
	hipMalloc((void**)&dB, size);
	hipMalloc((void**)&dC, size);
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
	
	dim3 dimBlock(TileSize, TileSize);
	dim3 dimGrid(nRows/TileSize, nRows/TileSize);

	
	Muldev<<<dimGrid, dimBlock, sizeof(double)*TileSize*TileSize*TileSize*TileSize>>>(dA, dB, dC, nRows);
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);
	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(incTime, startEvent_inc, stopEvent_inc);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return;//Placeholder
}


