#include<iostream>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RADIUS 3

int checkResults(int startElem, int endElem, float* cudaRes, float* res)
{
    int nDiffs=0;
    const float smallVal = 0.0001f;
    for(int i=startElem; i<endElem; i++)
        if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
    return nDiffs;
}

void initializeWeights(float* weights, int rad)
{
    // for now hardcoded for RADIUS=3
    weights[0] = 0.50f;
    weights[1] = 0.75f;
    weights[2] = 1.25f;
    weights[3] = 2.00f;
    weights[4] = 1.25f;
    weights[5] = 0.75f;
    weights[6] = 0.50f;
}

void initializeArray(FILE* fp,float* arr, int nElements)
{
    for( int i=0; i<nElements; i++){
        	int r=fscanf(fp,"%f",&arr[i]);
		if(r == EOF){
		  rewind(fp);
		}
    }
}

void applyStencil1D_SEQ(int sIdx, int eIdx, const float *weights, float *in, float *out) {
  
  for (int i = sIdx; i < eIdx; i++) {   
    out[i] = 0;
    //loop over all elements in the stencil
    for (int j = -RADIUS; j <= RADIUS; j++) {
      out[i] += weights[j + RADIUS] * in[i + j]; 
    }
    out[i] = out[i] / (2 * RADIUS + 1);
  }
}

__global__ void applyStencil1D(int sIdx, int eIdx, const float *weights, float *in, float *out) {
   // int i = sIdx + blockIdx.x*blockDim.x + threadIdx.x;
    __shared__ volatile float sharedInput[512];


    int sInd = threadIdx.x;
    int bInd = threadIdx.x+blockIdx.x*blockDim.x;

    if(bInd < sIdx+eIdx)
    sharedInput[sInd] = in[bInd];

    __syncthreads();
    __threadfence_block();


if((bInd-sIdx) < eIdx){
		    
		float result = 0.f;
		if(sInd < 509 && sInd > 2){
		 	
       			result += weights[0]*sharedInput[sInd-3];
        		result += weights[1]*sharedInput[sInd-2];
        		result += weights[2]*sharedInput[sInd-1];
        		result += weights[3]*sharedInput[sInd];
       	 		result += weights[4]*sharedInput[sInd+1];
        		result += weights[5]*sharedInput[sInd+2];
        		result += weights[6]*sharedInput[sInd+3];
			result /=7.f;
   			out[bInd] = result;
			}
		if(sInd == 509){
       		 	result += weights[0]*sharedInput[sInd-3];
        		result += weights[1]*sharedInput[sInd-2];
        		result += weights[2]*sharedInput[sInd-1];
        		result += weights[3]*sharedInput[sInd];
       	 		result += weights[4]*sharedInput[sInd+1];
        		result += weights[5]*sharedInput[sInd+2];
        		result += weights[6]*in[bInd+3];
			result /=7.f;
   			out[bInd] = result;
			}
		if(sInd == 510){
	       		 result += weights[0]*sharedInput[sInd-3];
        		result += weights[1]*sharedInput[sInd-2];
        		result += weights[2]*sharedInput[sInd-1];
        		result += weights[3]*sharedInput[sInd];
       	 		result += weights[4]*sharedInput[sInd+1];
        		result += weights[5]*in[bInd+2];
        		result += weights[6]*in[bInd+3];
			result /=7.f;
   			out[bInd] = result;
			}
		if(sInd == 511){
			result += weights[0]*sharedInput[sInd-3];
        		result += weights[1]*sharedInput[sInd-2];
        		result += weights[2]*sharedInput[sInd-1];
        		result += weights[3]*sharedInput[sInd];
       	 		result += weights[4]*in[bInd+1];
        		result += weights[5]*in[bInd+2];
        		result += weights[6]*in[bInd+3];
			result /=7.f;
   			out[bInd] = result;
			}	
		if(blockIdx.x > 0){
			if(sInd == 2){
				result += weights[0]*in[bInd-3];
        			result += weights[1]*sharedInput[sInd-2];
        			result += weights[2]*sharedInput[sInd-1];
        			result += weights[3]*sharedInput[sInd];
       	 			result += weights[4]*sharedInput[sInd+1];
        			result += weights[5]*sharedInput[sInd+2];
        			result += weights[6]*sharedInput[sInd+3];
				result /=7.f;
   				out[bInd] = result;
				}
			if(sInd == 1){				
				result += weights[0]*in[bInd-3];
        			result += weights[1]*in[bInd-2];
        			result += weights[2]*sharedInput[sInd-1];
        			result += weights[3]*sharedInput[sInd];
       	 			result += weights[4]*sharedInput[sInd+1];
        			result += weights[5]*sharedInput[sInd+2];
        			result += weights[6]*sharedInput[sInd+3];
				result /=7.f;
   				out[bInd] = result;
				}
			if(sInd == 0){
				result += weights[0]*in[bInd-3];
        			result += weights[1]*in[bInd-2];
        			result += weights[2]*in[bInd-1];
        			result += weights[3]*sharedInput[sInd];
       	 			result += weights[4]*sharedInput[sInd+1];
        			result += weights[5]*sharedInput[sInd+2];
        			result += weights[6]*sharedInput[sInd+3];
				result /=7.f;
   				out[bInd] = result;
				}
		}

		
   }
   /* if( i < eIdx ) {
        float result = 0.f;
        result += weights[0]*in[i-3];
        result += weights[1]*in[i-2];
        result += weights[2]*in[i-1];
        result += weights[3]*in[i];
        result += weights[4]*in[i+1];
        result += weights[5]*in[i+2];
        result += weights[6]*in[i+3];
        result /=7.f;
        out[i] = result;
    }*/
}

int main(int argc, char* argv[]) {
  if(argc!=2){
	printf("Usage %s N\n",argv[0]);
	return 1;
  }
  int N=atoi(argv[1]);
  FILE *fp = fopen("problem1.inp","r");
  int size = N * sizeof(float); 
  int wsize = (2 * RADIUS + 1) * sizeof(float); 
  //allocate resources
  float *weights = (float *)malloc(wsize);
  float *in      = (float *)malloc(size);
  float *out     = (float *)malloc(size); 
  float *cuda_out= (float *)malloc(size);
  float time = 0.f;
  initializeWeights(weights, RADIUS);
  initializeArray(fp,in, N);
  float *d_weights;  hipMalloc(&d_weights, wsize);
  float *d_in;       hipMalloc(&d_in, size);
  float *d_out;      hipMalloc(&d_out, size);

	hipEvent_t startEvent_inc;
	hipEvent_t stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
	hipEventRecord(startEvent_inc,0);
  
  hipMemcpy(d_weights,weights,wsize,hipMemcpyHostToDevice);
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  applyStencil1D<<<(N+511)/512, 512, 512*sizeof(float)>>>(RADIUS, N-RADIUS, d_weights, d_in, d_out);
  hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);

	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&time, startEvent_inc, stopEvent_inc);



  applyStencil1D_SEQ(RADIUS, N-RADIUS, weights, in, out);
  int nDiffs = checkResults(RADIUS, N-RADIUS, cuda_out, out);

  if(nDiffs)printf("Test Failed\n"); // This should never print
  printf("%f\n%f\n",cuda_out[N-RADIUS-1],time);

  
  //free resources 
  free(weights); free(in); free(out); free(cuda_out);
  hipFree(d_weights);  hipFree(d_in);  hipFree(d_out);
  return 0;
}
