#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void addKernel(double* arrA, double* arrB, double* arrC, int N){
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	if(thid < N)
	arrC[thid] = arrA[thid]+arrB[thid];
}


int main( int argc, char *argv[])
{

	if(argc!=3)
	{
		printf("Invalid argument Usage: ./problem3 N M");
		return 0;
	}


	FILE *fpA,*fpB;
	int N = atoi(argv[1]);
	int M = atoi(argv[2]); 
	double *hA= new double[N];
	double *hB= new double[N];
	double *hC=  new double[N];
	double *refC=  new double[N]; // Used to verify functional correctness
	double *dA,*dB,*dC;  // You may use these to allocate memory on gpu
	//defining variables for timing
	hipEvent_t startEvent_inc, stopEvent_inc, startEvent_exc, stopEvent_exc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
	hipEventCreate(&startEvent_exc);
	hipEventCreate(&stopEvent_exc);
	float elapsedTime_inc, elapsedTime_exc;

	//reading files
	fpA = fopen("inputA.inp", "r");
	fpB= fopen("inputB.inp", "r");


	for (int i=0;i<N;i++){    
		fscanf(fpA, "%lf",&hA[i]);
	}
	for (int i=0;i<N;i++){
		fscanf(fpB, "%lf",&hB[i]);
	}



	int nBlocks = N/M;
	float blockRem = N/M - nBlocks;
	if(blockRem != 0)
	nBlocks = nBlocks+1;
	cout << "blocks used: " << nBlocks << "\n";



      for(int i=0;i<N;i++)
        refC[i]=hA[i]+hB[i];


	hipEventRecord(startEvent_inc,0);

	
 // starting timing for inclusive
	// TODO allocate memory for arrays and copay array A and B


	hipMalloc((void**)&dA, sizeof(double)*N);

	hipMemcpy(dA, hA, sizeof(double)*N, hipMemcpyHostToDevice);

	hipMalloc((void**)&dB, sizeof(double)*N);

	hipMemcpy(dB, hB, sizeof(double)*N, hipMemcpyHostToDevice);

	hipMalloc((void**)&dC, sizeof(double)*N);

	//cudaMemset(dC, 1, sizeof(double)*N);


	hipMemcpy(hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost);

	//cout << "first value of host array after copying back dC is: " << hC[0] << "\n";
	

	hipEventRecord(startEvent_exc,0); // staring timing for exclusive



	addKernel<<<nBlocks, M>>>(dA, dB, dC, N);



	//cout << "ran kernel" << "\n";
	hipEventRecord(stopEvent_exc,0);  // ending timing for exclusive
	hipEventSynchronize(stopEvent_exc);   
	hipEventElapsedTime(&elapsedTime_exc, startEvent_exc, stopEvent_exc);

	// TODO copy data back
	hipMemcpy(hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost);

	//cout << "last dC = " << dC[1] << "\n";

	//cout << "copied mem back" << "\n";
	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&elapsedTime_inc, startEvent_inc, stopEvent_inc);



	//verification
	int count=0;
	for(int i=0;i<N;i++)
	{
		//cout << hC[i]  << "\n";
		//cout << hA[i] << "\n";
		if(hC[i]!=refC[i])
		{
			count++;
		}
	}
	if(count!=0) // This should never be printed in correct code
		cout<<"Error at "<< count<<" locations\n";

	cout << "time to print results: " << "\n";
	cout<<N<<"\n";
	cout<<M<<"\n";
	cout <<elapsedTime_exc<<"\n";
	cout <<elapsedTime_inc<<"\n";
	cout <<hC[N-1]<<"\n";
	//freeing memory
	delete[] hA,hB,hC,refC;     

	// TODO free CUDA memory allocated
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return 0;
}



	
	






