#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void addKernel(double* arrA, double* arrB, double* arrC, int N){
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	if(thid < N)
	arrC[thid] += arrA[thid]+arrB[thid];
}


int main( int argc, char *argv[])
{

	if(argc!=3)
	{
		printf("Invalid argument Usage: ./problem3 N M");
		return 0;
	}

	cout << "HEre we go!" << "\n";

	FILE *fpA,*fpB;
	int N = atoi(argv[1]);
	int M = atoi(argv[2]); 
	double *hA= new double[N];
	double *hB= new double[N];
	double *hC=  new double[N];
	double *refC=  new double[N]; // Used to verify functional correctness
	double *dA,*dB,*dC;  // You may use these to allocate memory on gpu
	//defining variables for timing
	hipEvent_t startEvent_inc, stopEvent_inc, startEvent_exc, stopEvent_exc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
	hipEventCreate(&startEvent_exc);
	hipEventCreate(&stopEvent_exc);
	float elapsedTime_inc, elapsedTime_exc;

	//reading files
	fpA = fopen("inputA.inp", "r");
	fpB= fopen("inputB.inp", "r");


	for (int i=0;i<N;i++){    
		fscanf(fpA, "%lf",&hA[i]);
	}
	for (int i=0;i<N;i++){
		fscanf(fpB, "%lf",&hB[i]);
	}



	int nBlocks = N/M;
	float blockRem = N/M - nBlocks;
	if(blockRem != 0)
	nBlocks = nBlocks+1;
	cout << "blocks used: " << nBlocks << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";


      for(int i=0;i<N;i++)
        refC[i]=hA[i]+hB[i];

	cout << "starting cuda stuff" << "\n";
	cout << "right before record" << "\n";
	hipEventRecord(startEvent_inc,0);

	cout << "right after record"  << "\n";
 // starting timing for inclusive
	// TODO allocate memory for arrays and copay array A and B
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";

	hipMalloc((void**)&dA, sizeof(double)*N);
	cout << "dA allocated" << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	hipMemcpy(&dA, hA, sizeof(double)*N, hipMemcpyHostToDevice);
	cout << "dA copied " << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	hipMalloc((void**)&dB, sizeof(double)*N);
	cout << "dB allocated" << "\n";
	hipMemcpy(&dB, hB, sizeof(double)*N, hipMemcpyHostToDevice);
	cout << "dB copied " << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	hipMalloc((void**)&dC, sizeof(double)*N);
	cout << "dC copied " << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	//cudaMemset(dC, 1, sizeof(double)*N);

	cout << "trying to copy hC to dC" << "\n";
	hipMemcpy(hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost);
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	cout << "copied hC to dC " << "\n";
	//cout << "first value of host array after copying back dC is: " << hC[0] << "\n";
	
	cout << "alocated memory" << "\n";
	hipEventRecord(startEvent_exc,0); // staring timing for exclusive


	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";

	addKernel<<<nBlocks, M>>>(dA, dB, dC, N);


	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";

	cout << "ran kernel" << "\n";
	hipEventRecord(stopEvent_exc,0);  // ending timing for exclusive
	hipEventSynchronize(stopEvent_exc);   
	hipEventElapsedTime(&elapsedTime_exc, startEvent_exc, stopEvent_exc);

	// TODO copy data back
	hipMemcpy(hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost);

	cout << "copied mem back" << "\n";
	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&elapsedTime_inc, startEvent_inc, stopEvent_inc);


/*
	//verification
	int count=0;
	for(int i=0;i<N;i++)
	{
		//cout << hC[i]  << "\n";
		//cout << hA[i] << "\n";
		if(hC[i]!=refC[i])
		{
			count++;
		}
	}
	if(count!=0) // This should never be printed in correct code
		cout<<"Error at "<< count<<" locations\n";
*/
	cout << "time to print results: " << "\n";
	cout<<N<<"\n";
	cout<<M<<"\n";
	cout <<elapsedTime_exc<<"\n";
	cout <<elapsedTime_inc<<"\n";
	cout <<hC[N-1]<<"\n";
	//freeing memory
	delete[] hA,hB,hC,refC;     

	// TODO free CUDA memory allocated
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return 0;
}



	
	






