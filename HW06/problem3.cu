#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void addKernel(double* arrA, double* arrB, double* arrC, int N){
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	if(thid < N)
	arrC[thid] = arrA[thid]+arrB[thid];
}


int main( int argc, char *argv[])
{

	if(argc!=3)
	{
		printf("Invalid argument Usage: ./problem3 N M");
		return 0;
	}

	FILE *fpA,*fpB;
	int N = atoi(argv[1]);
	int M = atoi(argv[2]); 
	double *hA= new double[N];
	double *hB= new double[N];
	double *hC=  new double[N];
	double *refC=  new double[N]; // Used to verify functional correctness
	double *dA,*dB,*dC;  // You may use these to allocate memory on gpu
	//defining variables for timing
	hipEvent_t startEvent_inc, stopEvent_inc, startEvent_exc, stopEvent_exc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
	hipEventCreate(&startEvent_exc);
	hipEventCreate(&stopEvent_exc);
	float elapsedTime_inc, elapsedTime_exc;

	//reading files
	fpA = fopen("inputA.inp", "r");
	fpB= fopen("inputB.inp", "r");


	for (int i=0;i<N;i++){    
		fscanf(fpA, "%lf",&hA[i]);
	}
	for (int i=0;i<N;i++){
		fscanf(fpB, "%lf",&hB[i]);
	}



	int nBlocks = N/M;
	float blockRem = N/M - nBlocks;
	if(blockRem != 0)
	nBlocks = nBlocks+1;


      for(int i=0;i<N;i++)
        refC[i]=hA[i]+hB[i];

	cout << "starting cuda stuff" << "\n";
	hipEventRecord(startEvent_inc,0); // starting timing for inclusive
	// TODO allocate memory for arrays and copay array A and B
	hipMalloc((void**)&dA, sizeof(double)*N);
	hipMemcpy(&dA, hA, sizeof(double)*N, hipMemcpyHostToDevice);
	hipMalloc((void**)&dB, sizeof(double)*N);
	hipMemcpy(&dB, hB, sizeof(double)*N, hipMemcpyHostToDevice);
	hipMalloc((void**)&dC, sizeof(double)*N);
	//cudaMemset(dB, 0, sizeof(double)*N);
	
	cout << "alocated memory" << "\n";
	hipEventRecord(startEvent_exc,0); // staring timing for exclusive

	addKernel<<<nBlocks, M>>>(dA, dB, dC, N);

	cout << "ran kernel" << "\n";
	hipEventRecord(stopEvent_exc,0);  // ending timing for exclusive
	hipEventSynchronize(stopEvent_exc);   
	hipEventElapsedTime(&elapsedTime_exc, startEvent_exc, stopEvent_exc);

	// TODO copy data back
	hipMemcpy(&hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost);

	cout << "copied mem back" << "\n";
	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&elapsedTime_inc, startEvent_inc, stopEvent_inc);



	//verification
	int count=0;
	for(int i=0;i<N;i++)
	{
		if(hC[i]!=refC[i])
		{
			count++;
		}
	}
	if(count!=0) // This should never be printed in correct code
		std::cout<<"Error at "<< count<<" locations\n";
	std::cout<<N<<"\n"<<M<<"\n"<<elapsedTime_exc<<"\n"<<elapsedTime_inc<<"\n"<<hC[N-1]<<"\n";
	//freeing memory
	delete[] hA,hB,hC,refC;     

	// TODO free CUDA memory allocated
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return 0;
}



	
	






