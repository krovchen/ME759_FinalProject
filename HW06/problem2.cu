#include<hip/hip_runtime.h>
#include<iostream>

using namespace std;


__global__ void simpleKernel(int* data, int size){
//this adds a value to a variable stored in global memory
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	if(thid < size)
	data[thid] = (blockIdx.x+ threadIdx.x);
}

int main()
{
	const int numElems = 16;
	int hostArray[numElems], *dArray;
	int i = 0;
	hipMalloc((void**)&dArray, sizeof(int)*numElems);
	hipMemset(dArray, 0, numElems*sizeof(int));
	
	simpleKernel<<<2, 8>>>(dArray, numElems);
	hipMemcpy(&hostArray, dArray, sizeof(int)*numElems, hipMemcpyDeviceToHost);

	cout << "Values in hostArray: " << endl;
	for(i = 0; i < numElems; i++)
		cout << hostArray[i] << endl;
	hipFree(dArray);
	return 0;
}
