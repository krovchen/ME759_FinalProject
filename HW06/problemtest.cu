#include<hip/hip_runtime.h>
#include<iostream>

using namespace std;


__global__ void simpleKernel(int* data){
//this adds a value to a variable stored in global memory
	//int thid = threadIdx.x+blockIdx.x*blockDim.x;
	//if(thid < size)
	data[threadIdx.x] += 2*(blockIdx.x+threadIdx.x);
}

int main()
{
	const int numElems = 4;
	int hostArray[numElems], *devArray;
	int testArray[numElems];
	int i = 0;

	hipMalloc((void**)&devArray, sizeof(int)*numElems);
	//set devArray to 0 all elements
	hipMemset(devArray, 0, numElems*sizeof(int));
	//expecting this to set testArray to devArray	
	hipMemcpy(&testArray, devArray, sizeof(int)*numElems, hipMemcpyDeviceToHost);

	//here run kernel. same as lecture slide
	simpleKernel<<<1, 4>>>(devArray);


	//now copy devArray to hostArray to get the same answer in lecture
	hipMemcpy(&hostArray, devArray, sizeof(int)*numElems, hipMemcpyDeviceToHost);

	cout << "Values in hostArray: " << endl;
	for(i = 0; i < numElems; i++)
		cout << hostArray[i] << endl;
	cout << "Values in testArray: " << endl;
	for(i = 0; i < numElems; i++)
		cout << testArray[i] << endl;
	hipFree(devArray);


	return 0;
}
