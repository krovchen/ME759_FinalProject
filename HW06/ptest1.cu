#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void addKernel(double* arrA, double* arrB, double* arrC, int N){
	int thid = threadIdx.x+blockIdx.x*blockDim.x;
	if(thid < N)
	arrC[thid] = threadIdx.x;
}


int main( int argc, char *argv[])
{

	if(argc!=3)
	{
		printf("Invalid argument Usage: ./problem3 N M");
		return 0;
	}

	cout << "HEre we go!" << "\n";


	int N = atoi(argv[1]);
	int M = atoi(argv[2]); 
	double *hA= new double[N];
	double *hB= new double[N];
	double *hC=  new double[N];
	double *refC=  new double[N]; // Used to verify functional correctness
	double *dA,*dB,*dC;  // You may use these to allocate memory on gpu
	//defining variables for timing


	int nBlocks = N/M;
	float blockRem = N/M - nBlocks;
	if(blockRem != 0)
	nBlocks = nBlocks+1;
	cout << "blocks used: " << nBlocks << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";


      for(int i=0;i<N;i++)
        refC[i]=hA[i]+hB[i];

	cout << "starting cuda stuff" << "\n";
	cout << "right before record" << "\n";


	cout << "right after record"  << "\n";
 // starting timing for inclusive
	// TODO allocate memory for arrays and copay array A and B
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";

	hipMalloc((void**)&dA, sizeof(double)*N);
	cout << "dA allocated" << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	hipMemcpy(&dA, hA, sizeof(double)*N, hipMemcpyHostToDevice);
	cout << "dA copied " << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	hipMalloc((void**)&dB, sizeof(double)*N);
	cout << "dB allocated" << "\n";
	hipMemcpy(&dB, hB, sizeof(double)*N, hipMemcpyHostToDevice);
	cout << "dB copied " << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	hipMalloc((void**)&dC, sizeof(double)*N);
	cout << "dC copied " << "\n";
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	//cudaMemset(dC, 1, sizeof(double)*N);

	cout << "trying to copy hC to dC" << "\n";
	hipMemcpy(hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost);
	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";
	cout << "copied hC to dC " << "\n";
	//cout << "first value of host array after copying back dC is: " << hC[0] << "\n";
	
	cout << "alocated memory" << "\n";



	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";

	addKernel<<<nBlocks, M>>>(dA, dB, dC, N);


	cout << "N = " << N << "\n";
	cout << "M = " << M << "\n";

	cout << "ran kernel" << "\n";

	// TODO copy data back
	hipMemcpy(hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost);



	cout << "time to print results: " << "\n";
	cout<<N<<"\n";
	cout<<M<<"\n";

	cout <<hC[N-1]<<"\n";
	//freeing memory
	delete[] hA,hB,hC,refC;     

	// TODO free CUDA memory allocated
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return 0;
}



	
	






